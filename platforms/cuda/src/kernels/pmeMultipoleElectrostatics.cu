#include "hip/hip_runtime.h"
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

typedef struct {
    real3 pos, force, torque, inducedDipole, sphericalDipole;
    real q;
    float thole, damp;
#ifdef INCLUDE_QUADRUPOLES
    real sphericalQuadrupole[5];
#endif
#ifdef INCLUDE_OCTOPOLES
    real sphericalOctopole[7];
#endif
} AtomData;

inline __device__ void loadAtomData(AtomData& data, int atom, const real4* __restrict__ posq, const real* __restrict__ sphericalDipole,
            const real* __restrict__ sphericalQuadrupole, const real* __restrict__ sphericalOctopole, const real* __restrict__ inducedDipole,
            const float2* __restrict__ dampingAndThole) {
    real4 atomPosq = posq[atom];
    data.pos = make_real3(atomPosq.x, atomPosq.y, atomPosq.z);
    data.q = atomPosq.w;
    data.sphericalDipole.x = sphericalDipole[atom*3];
    data.sphericalDipole.y = sphericalDipole[atom*3+1];
    data.sphericalDipole.z = sphericalDipole[atom*3+2];
#ifdef INCLUDE_QUADRUPOLES
    data.sphericalQuadrupole[0] = sphericalQuadrupole[atom*5];
    data.sphericalQuadrupole[1] = sphericalQuadrupole[atom*5+1];
    data.sphericalQuadrupole[2] = sphericalQuadrupole[atom*5+2];
    data.sphericalQuadrupole[3] = sphericalQuadrupole[atom*5+3];
    data.sphericalQuadrupole[4] = sphericalQuadrupole[atom*5+4];
#endif
#ifdef INCLUDE_OCTOPOLES
    data.sphericalOctopole[0] = sphericalOctopole[atom*7];
    data.sphericalOctopole[1] = sphericalOctopole[atom*7+1];
    data.sphericalOctopole[2] = sphericalOctopole[atom*7+2];
    data.sphericalOctopole[3] = sphericalOctopole[atom*7+3];
    data.sphericalOctopole[4] = sphericalOctopole[atom*7+4];
    data.sphericalOctopole[5] = sphericalOctopole[atom*7+5];
    data.sphericalOctopole[6] = sphericalOctopole[atom*7+6];
#endif
    data.inducedDipole.x = inducedDipole[atom*3];
    data.inducedDipole.y = inducedDipole[atom*3+1];
    data.inducedDipole.z = inducedDipole[atom*3+2];
    float2 temp = dampingAndThole[atom];
    data.damp = temp.x;
    data.thole = temp.y;
}

__device__ float computeMScaleFactor(uint2 covalent, int index) {
    int mask = 1<<index;
    bool x = (covalent.x & mask);
    bool y = (covalent.y & mask);
	return (x ? (y ? 0.0f : (float) SCALEFACTOR14): 1.0f);
}


__device__ void computeOneInteraction(AtomData& atom1, AtomData& atom2, bool hasExclusions, float pScale, float mScale, float forceFactor,
                                      mixed& energy, real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ) {
    // Compute the displacement.
    real3 delta;
    delta.x = atom2.pos.x - atom1.pos.x;
    delta.y = atom2.pos.y - atom1.pos.y;
    delta.z = atom2.pos.z - atom1.pos.z;
    APPLY_PERIODIC_TO_DELTA(delta)
    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
    if (r2 > CUTOFF_SQUARED)
        return;

    real rInv = RSQRT(r2);
    real r = r2*rInv;

    // Rotate the various dipoles and quadrupoles.

    real qiRotationMatrix[3][3];
    buildQIRotationMatrix(delta, rInv, qiRotationMatrix);

    real3 qiUindI = 0.5f*make_real3(qiRotationMatrix[0][1]*atom1.inducedDipole.x + qiRotationMatrix[0][2]*atom1.inducedDipole.y + qiRotationMatrix[0][0]*atom1.inducedDipole.z,
                                    qiRotationMatrix[1][1]*atom1.inducedDipole.x + qiRotationMatrix[1][2]*atom1.inducedDipole.y + qiRotationMatrix[1][0]*atom1.inducedDipole.z,
                                    qiRotationMatrix[2][1]*atom1.inducedDipole.x + qiRotationMatrix[2][2]*atom1.inducedDipole.y + qiRotationMatrix[2][0]*atom1.inducedDipole.z);
    real3 qiUindJ = 0.5f*make_real3(qiRotationMatrix[0][1]*atom2.inducedDipole.x + qiRotationMatrix[0][2]*atom2.inducedDipole.y + qiRotationMatrix[0][0]*atom2.inducedDipole.z,
                                    qiRotationMatrix[1][1]*atom2.inducedDipole.x + qiRotationMatrix[1][2]*atom2.inducedDipole.y + qiRotationMatrix[1][0]*atom2.inducedDipole.z,
                                    qiRotationMatrix[2][1]*atom2.inducedDipole.x + qiRotationMatrix[2][2]*atom2.inducedDipole.y + qiRotationMatrix[2][0]*atom2.inducedDipole.z);

    real3 rotatedDipole1 = rotateDipole(atom1.sphericalDipole, qiRotationMatrix);
    real3 rotatedDipole2 = rotateDipole(atom2.sphericalDipole, qiRotationMatrix);
    real rotatedQuadrupole1[] = {0, 0, 0, 0, 0};
    real rotatedQuadrupole2[] = {0, 0, 0, 0, 0};
#ifdef INCLUDE_QUADRUPOLES
    rotateQuadupoles(qiRotationMatrix, atom1.sphericalQuadrupole, atom2.sphericalQuadrupole, rotatedQuadrupole1, rotatedQuadrupole2);
#endif    
    real rotatedOctopole1[] = {0, 0, 0, 0, 0, 0, 0};
    real rotatedOctopole2[] = {0, 0, 0, 0, 0, 0, 0};
#ifdef INCLUDE_OCTOPOLES
    rotateOctopoles(qiRotationMatrix, atom1.sphericalOctopole, atom2.sphericalOctopole, rotatedOctopole1, rotatedOctopole2);
#endif    
    // The field derivatives at I due to permanent and induced moments on J, and vice-versa.
    // Also, their derivatives w.r.t. R, which are needed for force calculations
    real Vij[16], Vji[16], VjiR[16], VijR[16];
    // The field derivatives at I due to only permanent moments on J, and vice-versa.
    real Vijd[3], Vjid[3];
    real rInvVec[9], alphaRVec[10], bVec[6];

    // The rInvVec array is defined such that the ith element is R^-i, with the
    // dieleectric constant folded in, to avoid conversions later.
    rInvVec[1] = rInv;
    for (int i = 2; i < 9; ++i)
        rInvVec[i] = rInvVec[i-1] * rInv;

    // The alpharVec array is defined such that the ith element is (alpha R)^i,
    // where kappa (alpha in OpenMM parlance) is the Ewald attenuation parameter.
    real ralpha = EWALD_ALPHA*r;
    real exp2a = EXP(-(ralpha*ralpha));
#ifdef USE_DOUBLE_PRECISION
    const real erfAlphaR = erf(ralpha);
#else
    // This approximation for erfc is from Abramowitz and Stegun (1964) p. 299.  They cite the following as
    // the original source: C. Hastings, Jr., Approximations for Digital Computers (1955).  It has a maximum
    // error of 1.5e-7.

    const real t = RECIP(1.0f+0.3275911f*ralpha);
    const real erfAlphaR = 1-(0.254829592f+(-0.284496736f+(1.421413741f+(-1.453152027f+1.061405429f*t)*t)*t)*t)*t*exp2a;
#endif
    alphaRVec[1] = ralpha;
    for (int i = 2; i < 10; ++i)
        alphaRVec[i] = alphaRVec[i-1]*ralpha;
    real X = 2*exp2a/SQRT_PI;
    int doubleFactorial = 1, facCount = 1;
    real tmp = alphaRVec[1];
    bVec[1] = -erfAlphaR;
    for (int i = 2; i < 6; ++i) {
        bVec[i] = bVec[i-1] + tmp * X / (real)(doubleFactorial);
        facCount = facCount + 2;
        doubleFactorial = doubleFactorial * facCount;
        tmp *= 2*alphaRVec[2];
    }

    real dmp = fabs(atom1.damp*atom2.damp);
    real a = pScale == 0 ? atom1.thole + atom2.thole : DEFAULT_THOLE_WIDTH;
    real u = dmp > (real)1.0E-5 ? r/dmp : (real)1E6;
    real au = a*u;
    real expau = au < (real)50 ? exp(-au) : (real)0;
    au =  dmp > (real)1.0E-5 ? au : 0;
    real au2 = au*au;
    real au3 = au2*au;
    real au4 = au3*au;
    real au5 = au4*au;
    real au6 = au5*au;
    // Thole damping factors for energies
    real thole_c   = 1 - expau*(1 + au + au2/2);
    real thole_d0  = 1 - expau*(1 + au + au2/2 + au3/4);
    real thole_d1  = 1 - expau*(1 + au + au2/2);
    real thole_q0  = 1 - expau*(1 + au + au2/2 + au3/6 + au4/18);
    real thole_q1  = 1 - expau*(1 + au + au2/2 + au3/6);
    real thole_o0  = 1 - expau*(1 + au + au2/2 + au3/6 + au4/24 + au5/120);
    real thole_o1  = 1 - expau*(1 + au + au2/2 + au3/6 + au4/30);
    // Thole damping factors for derivatives
    real dthole_c  = 1 - expau*(1 + au + au2/2 + au3/4);
    real dthole_d0 = 1 - expau*(1 + au + au2/2 + au3/6 + au4/12);
    real dthole_d1 = 1 - expau*(1 + au + au2/2 + au3/6);
    real dthole_q0 = 1 - expau*(1 + au + au2/2 + au3/6 + au4/24 + au5/72);
    real dthole_q1 = 1 - expau*(1 + au + au2/2 + au3/6 + au4/24);
    real dthole_o0 = 1 - expau*(1 + au + au2/2 + au3/6 + au4/24 + au5/120 + au6/600);
    real dthole_o1 = 1 - expau*(1 + au + au2/2 + au3/6 + au4/25 + au5/150);

    // Now we compute the (attenuated) Coulomb operator and its derivatives, contracted with
    // permanent moments and induced dipoles.  Note that the coefficient of the permanent force
    // terms is half of the expected value; this is because we compute the interaction of I with
    // the sum of induced and permanent moments on J, as well as the interaction of J with I's
    // permanent and induced moments; doing so double counts the permanent-permanent interaction.
    real ePermCoef, dPermCoef, eUIndCoef, dUIndCoef;

    // C-C terms (m=0)
    ePermCoef = rInvVec[1]*(mScale + bVec[2] - alphaRVec[1]*X);
    dPermCoef = -0.5f*(mScale + bVec[2])*rInvVec[2];
    Vij[0]  = ePermCoef*atom2.q;
    Vji[0]  = ePermCoef*atom1.q;
    VijR[0] = dPermCoef*atom2.q;
    VjiR[0] = dPermCoef*atom1.q;

    // C-D and C-Uind terms (m=0)
    ePermCoef = rInvVec[2]*(mScale + bVec[2]);
    eUIndCoef = 2*rInvVec[2]*(pScale*thole_c + bVec[2]);
    dPermCoef = -rInvVec[3]*(mScale + bVec[2] + alphaRVec[3]*X);
    dUIndCoef = -4*rInvVec[3]*(pScale*dthole_c + bVec[2] + alphaRVec[3]*X);
    Vij[0]  += -(ePermCoef*rotatedDipole2.x + eUIndCoef*qiUindJ.x);
    Vji[1]   = -(ePermCoef*atom1.q);
    VijR[0] += -(dPermCoef*rotatedDipole2.x + dUIndCoef*qiUindJ.x);
    VjiR[1]  = -(dPermCoef*atom1.q);
    Vjid[0]  = -(eUIndCoef*atom1.q);
    // D-C and Uind-C terms (m=0)
    Vij[1]   = ePermCoef*atom2.q;
    Vji[0]  += ePermCoef*rotatedDipole1.x + eUIndCoef*qiUindI.x;
    VijR[1]  = dPermCoef*atom2.q;
    VjiR[0] += dPermCoef*rotatedDipole1.x + dUIndCoef*qiUindI.x;
    Vijd[0]  = eUIndCoef*atom2.q;

    // D-D and D-Uind terms (m=0)
    const real twoThirds = (real) 2/3;
    ePermCoef = -twoThirds*rInvVec[3]*(3*(mScale + bVec[3]) + alphaRVec[3]*X);
    eUIndCoef = -2*twoThirds*rInvVec[3]*(3*(pScale*thole_d0 + bVec[3]) + alphaRVec[3]*X);
    dPermCoef = rInvVec[4]*(3*(mScale + bVec[3]) + 2*alphaRVec[5]*X);
    dUIndCoef = 2*rInvVec[4]*(6*(pScale*dthole_d0 + bVec[3]) + 4*alphaRVec[5]*X);
    Vij[1]  += ePermCoef*rotatedDipole2.x + eUIndCoef*qiUindJ.x;
    Vji[1]  += ePermCoef*rotatedDipole1.x + eUIndCoef*qiUindI.x;
    VijR[1] += dPermCoef*rotatedDipole2.x + dUIndCoef*qiUindJ.x;
    VjiR[1] += dPermCoef*rotatedDipole1.x + dUIndCoef*qiUindI.x;
    Vijd[0] += eUIndCoef*rotatedDipole2.x;
    Vjid[0] += eUIndCoef*rotatedDipole1.x;
    // D-D and D-Uind terms (m=1)
    ePermCoef = rInvVec[3]*(mScale + bVec[3] - twoThirds*alphaRVec[3]*X);
    eUIndCoef = 2*rInvVec[3]*(pScale*thole_d1 + bVec[3] - twoThirds*alphaRVec[3]*X);
    dPermCoef = -1.5f*rInvVec[4]*(mScale + bVec[3]);
    dUIndCoef = -6*rInvVec[4]*(pScale*dthole_d1 + bVec[3]);
    Vij[2]  = ePermCoef*rotatedDipole2.y + eUIndCoef*qiUindJ.y;
    Vji[2]  = ePermCoef*rotatedDipole1.y + eUIndCoef*qiUindI.y;
    VijR[2] = dPermCoef*rotatedDipole2.y + dUIndCoef*qiUindJ.y;
    VjiR[2] = dPermCoef*rotatedDipole1.y + dUIndCoef*qiUindI.y;
    Vij[3]  = ePermCoef*rotatedDipole2.z + eUIndCoef*qiUindJ.z;
    Vji[3]  = ePermCoef*rotatedDipole1.z + eUIndCoef*qiUindI.z;
    VijR[3] = dPermCoef*rotatedDipole2.z + dUIndCoef*qiUindJ.z;
    VjiR[3] = dPermCoef*rotatedDipole1.z + dUIndCoef*qiUindI.z;
    Vijd[1] = eUIndCoef*rotatedDipole2.y;
    Vjid[1] = eUIndCoef*rotatedDipole1.y;
    Vijd[2] = eUIndCoef*rotatedDipole2.z;
    Vjid[2] = eUIndCoef*rotatedDipole1.z;

    // C-Q terms (m=0)
    ePermCoef = (mScale + bVec[3])*rInvVec[3];
    dPermCoef = -((real) 1/3)*rInvVec[4]*(4.5f*(mScale + bVec[3]) + 2*alphaRVec[5]*X);
    Vij[0]  += ePermCoef*rotatedQuadrupole2[0];
    Vji[4]   = ePermCoef*atom1.q;
    VijR[0] += dPermCoef*rotatedQuadrupole2[0];
    VjiR[4]  = dPermCoef*atom1.q;
    // Q-C terms (m=0)
    Vij[4]   = ePermCoef*atom2.q;
    Vji[0]  += ePermCoef*rotatedQuadrupole1[0];
    VijR[4]  = dPermCoef*atom2.q;
    VjiR[0] += dPermCoef*rotatedQuadrupole1[0];

    // D-Q and Uind-Q terms (m=0)
    const real fourThirds = (real) 4/3;
    ePermCoef = rInvVec[4]*(3*(mScale + bVec[3]) + fourThirds*alphaRVec[5]*X);
    eUIndCoef = 2*rInvVec[4]*(3*(pScale*thole_q0 + bVec[3]) + fourThirds*alphaRVec[5]*X);
    dPermCoef = -fourThirds*rInvVec[5]*(4.5f*(mScale + bVec[3]) + (1 + alphaRVec[2])*alphaRVec[5]*X);
    dUIndCoef = -2*fourThirds*rInvVec[5]*(9*(pScale*dthole_q0 + bVec[3]) + 2*(1 + alphaRVec[2])*alphaRVec[5]*X);
    Vij[1]  += ePermCoef*rotatedQuadrupole2[0];
    Vji[4]  += ePermCoef*rotatedDipole1.x + eUIndCoef*qiUindI.x;
    VijR[1] += dPermCoef*rotatedQuadrupole2[0];
    VjiR[4] += dPermCoef*rotatedDipole1.x + dUIndCoef*qiUindI.x;
    Vijd[0] += eUIndCoef*rotatedQuadrupole2[0];
    // Q-D and Q-Uind terms (m=0)
    Vij[4]  += -(ePermCoef*rotatedDipole2.x + eUIndCoef*qiUindJ.x);
    Vji[1]  += -(ePermCoef*rotatedQuadrupole1[0]);
    VijR[4] += -(dPermCoef*rotatedDipole2.x + dUIndCoef*qiUindJ.x);
    VjiR[1] += -(dPermCoef*rotatedQuadrupole1[0]);
    Vjid[0] += -(eUIndCoef*rotatedQuadrupole1[0]);

    // D-Q and Uind-Q terms (m=1)
    const real sqrtThree = SQRT((real) 3);
    ePermCoef = -sqrtThree*rInvVec[4]*(mScale + bVec[3]);
    eUIndCoef = -2*sqrtThree*rInvVec[4]*(pScale*thole_q1 + bVec[3]);
    const real fourSqrtOneThird = 4/sqrt((real) 3);
    dPermCoef = fourSqrtOneThird*rInvVec[5]*(1.5f*(mScale + bVec[3]) + 0.5f*alphaRVec[5]*X);
    dUIndCoef = 2*fourSqrtOneThird*rInvVec[5]*(3*(pScale*dthole_q1 + bVec[3]) + alphaRVec[5]*X);
    Vij[2]  += ePermCoef*rotatedQuadrupole2[1];
    Vji[5]   = ePermCoef*rotatedDipole1.y + eUIndCoef*qiUindI.y;
    VijR[2] += dPermCoef*rotatedQuadrupole2[1];
    VjiR[5]  = dPermCoef*rotatedDipole1.y + dUIndCoef*qiUindI.y;
    Vij[3]  += ePermCoef*rotatedQuadrupole2[2];
    Vji[6]   = ePermCoef*rotatedDipole1.z + eUIndCoef*qiUindI.z;
    VijR[3] += dPermCoef*rotatedQuadrupole2[2];
    VjiR[6]  = dPermCoef*rotatedDipole1.z + dUIndCoef*qiUindI.z;
    Vijd[1] += eUIndCoef*rotatedQuadrupole2[1];
    Vijd[2] += eUIndCoef*rotatedQuadrupole2[2];
    // D-Q and Uind-Q terms (m=1)
    Vij[5]   = -(ePermCoef*rotatedDipole2.y + eUIndCoef*qiUindJ.y);
    Vji[2]  += -(ePermCoef*rotatedQuadrupole1[1]);
    VijR[5]  = -(dPermCoef*rotatedDipole2.y + dUIndCoef*qiUindJ.y);
    VjiR[2] += -(dPermCoef*rotatedQuadrupole1[1]);
    Vij[6]   = -(ePermCoef*rotatedDipole2.z + eUIndCoef*qiUindJ.z);
    Vji[3]  += -(ePermCoef*rotatedQuadrupole1[2]);
    VijR[6]  = -(dPermCoef*rotatedDipole2.z + dUIndCoef*qiUindJ.z);
    VjiR[3] += -(dPermCoef*rotatedQuadrupole1[2]);
    Vjid[1] += -(eUIndCoef*rotatedQuadrupole1[1]);
    Vjid[2] += -(eUIndCoef*rotatedQuadrupole1[2]);

    // Q-Q terms (m=0)
    ePermCoef = rInvVec[5]*(6*(mScale + bVec[4]) + ((real) 4/45)*(-3 + 10*alphaRVec[2])*alphaRVec[5]*X);
    dPermCoef = -rInvVec[6]*(135*(mScale + bVec[4]) + 4*(1 + 2*alphaRVec[2])*alphaRVec[7]*X)/9;
    Vij[4]  += ePermCoef*rotatedQuadrupole2[0];
    Vji[4]  += ePermCoef*rotatedQuadrupole1[0];
    VijR[4] += dPermCoef*rotatedQuadrupole2[0];
    VjiR[4] += dPermCoef*rotatedQuadrupole1[0];
    // Q-Q terms (m=1)
    const real fourOverFifteen = (real) 4/15;
    ePermCoef = -fourOverFifteen*rInvVec[5]*(15*(mScale + bVec[4]) + alphaRVec[5]*X);
    dPermCoef = rInvVec[6]*(10*(mScale + bVec[4]) + fourThirds*alphaRVec[7]*X);
    Vij[5]  += ePermCoef*rotatedQuadrupole2[1];
    Vji[5]  += ePermCoef*rotatedQuadrupole1[1];
    VijR[5] += dPermCoef*rotatedQuadrupole2[1];
    VjiR[5] += dPermCoef*rotatedQuadrupole1[1];
    Vij[6]  += ePermCoef*rotatedQuadrupole2[2];
    Vji[6]  += ePermCoef*rotatedQuadrupole1[2];
    VijR[6] += dPermCoef*rotatedQuadrupole2[2];
    VjiR[6] += dPermCoef*rotatedQuadrupole1[2];
    // Q-Q terms (m=2)
    ePermCoef = rInvVec[5]*(mScale + bVec[4] - fourOverFifteen*alphaRVec[5]*X);
    dPermCoef = -2.5f*(mScale + bVec[4])*rInvVec[6];
    Vij[7]  = ePermCoef*rotatedQuadrupole2[3];
    Vji[7]  = ePermCoef*rotatedQuadrupole1[3];
    VijR[7] = dPermCoef*rotatedQuadrupole2[3];
    VjiR[7] = dPermCoef*rotatedQuadrupole1[3];
    Vij[8]  = ePermCoef*rotatedQuadrupole2[4];
    Vji[8]  = ePermCoef*rotatedQuadrupole1[4];
    VijR[8] = dPermCoef*rotatedQuadrupole2[4];
    VjiR[8] = dPermCoef*rotatedQuadrupole1[4];

    // C-O (m=0)
    ePermCoef = rInvVec[4]*(-mScale - bVec[3] - fourOverFifteen*alphaRVec[5]*X);
    dPermCoef = 0.5f*fourOverFifteen*rInvVec[5]*(15*(mScale+bVec[3])+2*(2*alphaRVec[5]+alphaRVec[7])*X);
    Vij[0]  += ePermCoef*rotatedOctopole2[0];
    Vji[9]   = ePermCoef*atom1.q;
    VijR[0] += dPermCoef*rotatedOctopole2[0];
    VjiR[9]  = dPermCoef*atom1.q;
    // O-C (m=0)
    Vij[9]   = -ePermCoef*atom2.q;
    Vji[0]  -=  ePermCoef*rotatedOctopole1[0];
    VijR[9]  = -dPermCoef*atom2.q;
    VjiR[0] -=  dPermCoef*rotatedOctopole1[0];

    // D-O and Uind-O (m=0)
    const real twoOverFifteen = (real) 2/15;
    ePermCoef = -4*rInvVec[5]*(mScale+bVec[4]+twoOverFifteen*alphaRVec[7]*X);
    eUIndCoef = -8*rInvVec[5]*(pScale*thole_o0+bVec[4]+twoOverFifteen*alphaRVec[7]*X);
    dPermCoef = 0.5f*fourOverFifteen*rInvVec[6]*(75*(mScale+bVec[4])+4*(1+alphaRVec[2])*alphaRVec[7]*X);
    dUIndCoef = 2*fourOverFifteen*rInvVec[6]*(75*(pScale*dthole_o0+bVec[4])+4*(1+alphaRVec[2])*alphaRVec[7]*X);
    Vij[1]  += ePermCoef*rotatedOctopole2[0];
    Vji[9]  += ePermCoef*rotatedDipole1.x + eUIndCoef*qiUindI.x;
    VijR[1] += dPermCoef*rotatedOctopole2[0];
    VjiR[9] += dPermCoef*rotatedDipole1.x + dUIndCoef*qiUindI.x;
    // O-D and O-Uind (m=0)
    Vij[9]  += ePermCoef*rotatedDipole2.x + eUIndCoef*qiUindJ.x;
    Vji[1]  += ePermCoef*rotatedOctopole1[0];
    VijR[9] += dPermCoef*rotatedDipole2.x + dUIndCoef*qiUindJ.x;
    VjiR[1] += dPermCoef*rotatedOctopole1[0];
    Vijd[0] += eUIndCoef*rotatedOctopole2[0];
    Vjid[0] += eUIndCoef*rotatedOctopole1[0];
    // D-O and O-Uind (m=1)
    const real sqrtSix = SQRT((real) 6);
    const real sqrtTwoThirds = sqrt((real) 2/3);
    ePermCoef = sqrtSix*(mScale+bVec[4])*rInvVec[5];
    eUIndCoef = 2*sqrtSix*(pScale*thole_o1+bVec[4])*rInvVec[5];
    dPermCoef = -0.1f*sqrtTwoThirds*rInvVec[6]*(75*(mScale+bVec[4])+8*alphaRVec[7]*X);
    dUIndCoef = -0.4f*sqrtTwoThirds*rInvVec[6]*(75*(pScale*dthole_o1+bVec[4])+8*alphaRVec[7]*X);
    Vij[2]   += ePermCoef*rotatedOctopole2[1];
    Vji[10]   = ePermCoef*rotatedDipole1.y + eUIndCoef*qiUindI.y;
    VijR[2]  += dPermCoef*rotatedOctopole2[1];
    VjiR[10]  = dPermCoef*rotatedDipole1.y + dUIndCoef*qiUindI.y;
    Vij[3]   += ePermCoef*rotatedOctopole2[2];
    Vji[11]   = ePermCoef*rotatedDipole1.z + eUIndCoef*qiUindI.z;
    VijR[3]  += dPermCoef*rotatedOctopole2[2];
    VjiR[11]  = dPermCoef*rotatedDipole1.z + dUIndCoef*qiUindI.z;
    Vijd[1] += eUIndCoef*rotatedOctopole2[1];
    Vijd[2] += eUIndCoef*rotatedOctopole2[2];
    // O-D and O-Uind (m=1)
    Vij[10]   = ePermCoef*rotatedDipole2.y + eUIndCoef*qiUindJ.y;
    Vji[2]   += ePermCoef*rotatedOctopole1[1];
    VijR[10]  = dPermCoef*rotatedDipole2.y + dUIndCoef*qiUindJ.y;
    VjiR[2]  += dPermCoef*rotatedOctopole1[1];
    Vij[11]   = ePermCoef*rotatedDipole2.z + eUIndCoef*qiUindJ.z;
    Vji[3]   += ePermCoef*rotatedOctopole1[2];
    VijR[11]  = dPermCoef*rotatedDipole2.z + dUIndCoef*qiUindJ.z;
    VjiR[3]  += dPermCoef*rotatedOctopole1[2];
    Vjid[1] += eUIndCoef*rotatedOctopole1[1];
    Vjid[2] += eUIndCoef*rotatedOctopole1[2];

    // Q-O (m=0)
    const real fourOverFortyFive = (real) 4/45;
    ePermCoef = rInvVec[6]*(-10*(mScale+bVec[4]) - 2*fourOverFortyFive*(3+2*alphaRVec[2])*alphaRVec[7]*X);
    dPermCoef = 0.5f*fourOverFortyFive*rInvVec[7]*(675*(mScale+bVec[4])+2*(27+4*alphaRVec[4])*alphaRVec[7]*X);
    Vij[4]  += ePermCoef*rotatedOctopole2[0];
    Vji[9]  += ePermCoef*rotatedQuadrupole1[0];
    VijR[4] += dPermCoef*rotatedOctopole2[0];
    VjiR[9] += dPermCoef*rotatedQuadrupole1[0];
    // O-Q (m=0)
    Vij[9]  -= ePermCoef*rotatedQuadrupole2[0];
    Vji[4]  -= ePermCoef*rotatedOctopole1[0];
    VijR[9] -= dPermCoef*rotatedQuadrupole2[0];
    VjiR[4] -= dPermCoef*rotatedOctopole1[0];
    // Q-O (m=1)
    const real sqrtTwo = SQRT((real) 2);
    const real sqrtEightOverFifteen = SQRT((real) 8)/15;
    const real eightOverSeventyFive = (real) 8/75;
    ePermCoef = 5*sqrtTwo*rInvVec[6]*(mScale+bVec[4] + eightOverSeventyFive*alphaRVec[7]*X);
    dPermCoef = -0.5f*sqrtEightOverFifteen*rInvVec[7]*(225*(mScale+bVec[4])+8*(2+alphaRVec[2])*alphaRVec[7]*X);
    Vij[5]   += ePermCoef*rotatedOctopole2[1];
    Vji[10]  += ePermCoef*rotatedQuadrupole1[1];
    VijR[5]  += dPermCoef*rotatedOctopole2[1];
    VjiR[10] += dPermCoef*rotatedQuadrupole1[1];
    Vij[6]   += ePermCoef*rotatedOctopole2[2];
    Vji[11]  += ePermCoef*rotatedQuadrupole1[2];
    VijR[6]  += dPermCoef*rotatedOctopole2[2];
    VjiR[11] += dPermCoef*rotatedQuadrupole1[2];
    // O-Q (m=1)
    Vij[10]  -= ePermCoef*rotatedQuadrupole2[1];
    Vji[5]   -= ePermCoef*rotatedOctopole1[1];
    VijR[10] -= dPermCoef*rotatedQuadrupole2[1];
    VjiR[5]  -= dPermCoef*rotatedOctopole1[1];
    Vij[11]  -= ePermCoef*rotatedQuadrupole2[2];
    Vji[6]   -= ePermCoef*rotatedOctopole1[2];
    VijR[11] -= dPermCoef*rotatedQuadrupole2[2];
    VjiR[6]  -= dPermCoef*rotatedOctopole1[2];
    // Q-O (m=2)
    const real sqrtFive = SQRT((real) 5);
    const real twoSqrtFiveOverFifteen = 2*SQRT((real) 5)/15;
    ePermCoef = -sqrtFive*(mScale+bVec[4])*rInvVec[6];
    dPermCoef = 0.5f*twoSqrtFiveOverFifteen*rInvVec[7]*(45*(mScale+bVec[4])+4*alphaRVec[7]*X);
    Vij[7]  += ePermCoef*rotatedOctopole2[3];
    Vji[12]  = ePermCoef*rotatedQuadrupole1[3];
    VijR[7] += dPermCoef*rotatedOctopole2[3];
    VjiR[12] = dPermCoef*rotatedQuadrupole1[3];
    Vij[8]  += ePermCoef*rotatedOctopole2[4];
    Vji[13]  = ePermCoef*rotatedQuadrupole1[4];
    VijR[8] += dPermCoef*rotatedOctopole2[4];
    VjiR[13] = dPermCoef*rotatedQuadrupole1[4];
    // O-Q (m=2)
    Vij[12]  = -ePermCoef*rotatedQuadrupole2[3];
    Vji[7]  -=  ePermCoef*rotatedOctopole1[3];
    VijR[12] = -dPermCoef*rotatedQuadrupole2[3];
    VjiR[7] -=  dPermCoef*rotatedOctopole1[3];
    Vij[13]  = -ePermCoef*rotatedQuadrupole2[4];
    Vji[8]  -=  ePermCoef*rotatedOctopole1[4];
    VijR[13] = -dPermCoef*rotatedQuadrupole2[4];
    VjiR[8] -=  dPermCoef*rotatedOctopole1[4];

    // O-O (m=0)
    const real eightOverOneFiveSevenFive = (real) 8/1575;
    const real fourOverTwoTwoFive = (real) 4/225;
    ePermCoef = rInvVec[7]*(-20*(mScale+bVec[5]) - eightOverOneFiveSevenFive*(15+28*alphaRVec[2]+28*alphaRVec[4])*alphaRVec[7]*X);
    dPermCoef = 0.5f*fourOverTwoTwoFive*rInvVec[8]*(7875*(mScale+bVec[5])+4*(41 - 4*alphaRVec[2]+4*alphaRVec[4])*alphaRVec[9]*X);
    Vij[9]  += ePermCoef*rotatedOctopole2[0];
    Vji[9]  += ePermCoef*rotatedOctopole1[0];
    VijR[9] += dPermCoef*rotatedOctopole2[0];
    VjiR[9] += dPermCoef*rotatedOctopole1[0];
    // O-O (m=1)
    const real twoOverOneFiveZero = (real) 2/150;
    const real eightOverFiveTwoFive = (real) 8/525;
    ePermCoef = rInvVec[7]*(15*(mScale+bVec[5]) + eightOverFiveTwoFive*(-5 + 28*alphaRVec[2])* alphaRVec[7]*X);
    dPermCoef = -0.5f*twoOverOneFiveZero*rInvVec[8]*(7875*(mScale+bVec[5]) + 32*(3 + 2*alphaRVec[2])*alphaRVec[9]*X);
    Vij[10]  += ePermCoef*rotatedOctopole2[1];
    Vji[10]  += ePermCoef*rotatedOctopole1[1];
    VijR[10] += dPermCoef*rotatedOctopole2[1];
    VjiR[10] += dPermCoef*rotatedOctopole1[1];
    Vij[11]  += ePermCoef*rotatedOctopole2[2];
    Vji[11]  += ePermCoef*rotatedOctopole1[2];
    VijR[11] += dPermCoef*rotatedOctopole2[2];
    VjiR[11] += dPermCoef*rotatedOctopole1[2];
    // O-O (m=2)
    const real eightOverOneHundredFive = (real) 8/105;
    const real sixteenOverFifteen = (real) 16/15;
    ePermCoef = rInvVec[7]*(-6*(mScale+bVec[5]) - eightOverOneHundredFive*alphaRVec[7]*X);
    dPermCoef = 0.5f*rInvVec[8]*(42*(mScale+bVec[5]) + sixteenOverFifteen*alphaRVec[9]*X);
    Vij[12]  += ePermCoef*rotatedOctopole2[3];
    Vji[12]  += ePermCoef*rotatedOctopole1[3];
    VijR[12] += dPermCoef*rotatedOctopole2[3];
    VjiR[12] += dPermCoef*rotatedOctopole1[3];
    Vij[13]  += ePermCoef*rotatedOctopole2[4];
    Vji[13]  += ePermCoef*rotatedOctopole1[4];
    VijR[13] += dPermCoef*rotatedOctopole2[4];
    VjiR[13] += dPermCoef*rotatedOctopole1[4];
    // O-O (m=3)
    ePermCoef = rInvVec[7]*((mScale+bVec[5]) - eightOverOneHundredFive*alphaRVec[7]*X);
    dPermCoef = -0.5f*7*(mScale+bVec[5])*rInvVec[8];
    Vij[14]  = ePermCoef*rotatedOctopole2[5];
    Vji[14]  = ePermCoef*rotatedOctopole1[5];
    VijR[14] = dPermCoef*rotatedOctopole2[5];
    VjiR[14] = dPermCoef*rotatedOctopole1[5];
    Vij[15]  = ePermCoef*rotatedOctopole2[6];
    Vji[15]  = ePermCoef*rotatedOctopole1[6];
    VijR[15] = dPermCoef*rotatedOctopole2[6];
    VjiR[15] = dPermCoef*rotatedOctopole1[6];

    // Evaluate the energies, forces and torques due to permanent+induced moments
    // interacting with just the permanent moments.
    energy += forceFactor*0.5f*(
        atom1.q*Vij[0]
              + rotatedDipole1.x*Vij[1] + rotatedDipole1.y*Vij[2] + rotatedDipole1.z*Vij[3]
              + rotatedQuadrupole1[0]*Vij[4] + rotatedQuadrupole1[1]*Vij[5] + rotatedQuadrupole1[2]*Vij[6] + rotatedQuadrupole1[3]*Vij[7] + rotatedQuadrupole1[4]*Vij[8]
              + rotatedOctopole1[0]*Vij[9] + rotatedOctopole1[1]*Vij[10] + rotatedOctopole1[2]*Vij[11] + rotatedOctopole1[3]*Vij[12]
              + rotatedOctopole1[4]*Vij[13] + rotatedOctopole1[5]*Vij[14] + rotatedOctopole1[6]*Vij[15]
      + atom2.q*Vji[0]
              + rotatedDipole2.x*Vji[1] + rotatedDipole2.y*Vji[2] + rotatedDipole2.z*Vji[3]
              + rotatedQuadrupole2[0]*Vji[4] + rotatedQuadrupole2[1]*Vji[5] + rotatedQuadrupole2[2]*Vji[6] + rotatedQuadrupole2[3]*Vji[7] + rotatedQuadrupole2[4]*Vji[8]
              + rotatedOctopole2[0]*Vji[9] + rotatedOctopole2[1]*Vji[10] + rotatedOctopole2[2]*Vji[11] + rotatedOctopole2[3]*Vji[12]
              + rotatedOctopole2[4]*Vji[13] + rotatedOctopole2[5]*Vji[14] + rotatedOctopole2[6]*Vji[15]
    );
    real fIZ = atom1.q*VijR[0]
             + rotatedDipole1.x*VijR[1] + rotatedDipole1.y*VijR[2] + rotatedDipole1.z*VijR[3]
             + rotatedQuadrupole1[0]*VijR[4] + rotatedQuadrupole1[1]*VijR[5] + rotatedQuadrupole1[2]*VijR[6] + rotatedQuadrupole1[3]*VijR[7] + rotatedQuadrupole1[4]*VijR[8]
             + rotatedOctopole1[0]*VijR[9] + rotatedOctopole1[1]*VijR[10] + rotatedOctopole1[2]*VijR[11] + rotatedOctopole1[3]*VijR[12]
             + rotatedOctopole1[4]*VijR[13] + rotatedOctopole1[5]*VijR[14] + rotatedOctopole1[6]*VijR[15]
             ;
    real fJZ = atom2.q*VjiR[0]
             + rotatedDipole2.x*VjiR[1] + rotatedDipole2.y*VjiR[2] + rotatedDipole2.z*VjiR[3]
             + rotatedQuadrupole2[0]*VjiR[4] + rotatedQuadrupole2[1]*VjiR[5] + rotatedQuadrupole2[2]*VjiR[6] + rotatedQuadrupole2[3]*VjiR[7] + rotatedQuadrupole2[4]*VjiR[8]
             + rotatedOctopole2[0]*VjiR[9] + rotatedOctopole2[1]*VjiR[10] + rotatedOctopole2[2]*VjiR[11] + rotatedOctopole2[3]*VjiR[12]
             + rotatedOctopole2[4]*VjiR[13] + rotatedOctopole2[5]*VjiR[14] + rotatedOctopole2[6]*VjiR[15]
             ;

    const real sqrtThreeHalves = SQRT((real)1.5f);
    const real sqrtFiveHalves = SQRT((real)2.5f);
    real EIX = 0 
             + rotatedDipole1.z*Vij[1]
             - rotatedDipole1.x*Vij[3]
             + sqrtThree*rotatedQuadrupole1[2]*Vij[4]
             + rotatedQuadrupole1[4]*Vij[5]
             - (sqrtThree*rotatedQuadrupole1[0]+rotatedQuadrupole1[3])*Vij[6]
             + rotatedQuadrupole1[2]*Vij[7]
             - rotatedQuadrupole1[1]*Vij[8]
             + sqrtSix*rotatedOctopole1[2]*Vij[9]
             + sqrtFiveHalves*rotatedOctopole1[4]*Vij[10]
             - (sqrtSix*rotatedOctopole1[0]+sqrtFiveHalves*rotatedOctopole1[3])*Vij[11]
             + (sqrtFiveHalves*rotatedOctopole1[2]+sqrtThreeHalves*rotatedOctopole1[6])*Vij[12]
             - (sqrtFiveHalves*rotatedOctopole1[1]+sqrtThreeHalves*rotatedOctopole1[5])*Vij[13]
             + sqrtThreeHalves*rotatedOctopole1[4]*Vij[14]
             - sqrtThreeHalves*rotatedOctopole1[3]*Vij[15]
             ;
    real EIY = 0
             - rotatedDipole1.y*Vij[1]
             + rotatedDipole1.x*Vij[2]
             - sqrtThree*rotatedQuadrupole1[1]*Vij[4]
             + (sqrtThree*rotatedQuadrupole1[0]-rotatedQuadrupole1[3])*Vij[5]
             - rotatedQuadrupole1[4]*Vij[6] 
             + rotatedQuadrupole1[1]*Vij[7] 
             + rotatedQuadrupole1[2]*Vij[8]
             - sqrtSix*rotatedOctopole1[1]*Vij[9]
             + (sqrtSix*rotatedOctopole1[0]-sqrtFiveHalves*rotatedOctopole1[3])*Vij[10]
             - sqrtFiveHalves*rotatedOctopole1[4]*Vij[11]
             + (sqrtFiveHalves*rotatedOctopole1[1]-sqrtThreeHalves*rotatedOctopole1[5])*Vij[12]
             + (sqrtFiveHalves*rotatedOctopole1[2]-sqrtThreeHalves*rotatedOctopole1[6])*Vij[13]
             + sqrtThreeHalves*rotatedOctopole1[3]*Vij[14]
             + sqrtThreeHalves*rotatedOctopole1[4]*Vij[15]
             ;
    real EIZ = 0
             - rotatedDipole1.z*Vij[2]
             + rotatedDipole1.y*Vij[3] 
             - rotatedQuadrupole1[2]*Vij[5] 
             + rotatedQuadrupole1[1]*Vij[6] 
             - 2*rotatedQuadrupole1[4]*Vij[7] 
             + 2*rotatedQuadrupole1[3]*Vij[8]
             - rotatedOctopole1[2]*Vij[10]
             + rotatedOctopole1[1]*Vij[11]
             - 2*rotatedOctopole1[4]*Vij[12]
             + 2*rotatedOctopole1[3]*Vij[13]
             - 3*rotatedOctopole1[6]*Vij[14]
             + 3*rotatedOctopole1[5]*Vij[15]
             ;
    real EJX = 0
             + rotatedDipole2.z*Vji[1]
             - rotatedDipole2.x*Vji[3]
             + sqrtThree*rotatedQuadrupole2[2]*Vji[4]
             + rotatedQuadrupole2[4]*Vji[5]
             - (sqrtThree*rotatedQuadrupole2[0]+rotatedQuadrupole2[3])*Vji[6]
             + rotatedQuadrupole2[2]*Vji[7]
             - rotatedQuadrupole2[1]*Vji[8]
             + sqrtSix*rotatedOctopole2[2]*Vji[9]
             + sqrtFiveHalves*rotatedOctopole2[4]*Vji[10]
             - (sqrtSix*rotatedOctopole2[0]+sqrtFiveHalves*rotatedOctopole2[3])*Vji[11]
             + (sqrtFiveHalves*rotatedOctopole2[2]+sqrtThreeHalves*rotatedOctopole2[6])*Vji[12]
             - (sqrtFiveHalves*rotatedOctopole2[1]+sqrtThreeHalves*rotatedOctopole2[5])*Vji[13]
             + sqrtThreeHalves*rotatedOctopole2[4]*Vji[14]
             - sqrtThreeHalves*rotatedOctopole2[3]*Vji[15]
             ;
    real EJY = 0
             - rotatedDipole2.y*Vji[1]
             + rotatedDipole2.x*Vji[2]
             - sqrtThree*rotatedQuadrupole2[1]*Vji[4]
             + (sqrtThree*rotatedQuadrupole2[0]-rotatedQuadrupole2[3])*Vji[5]
             - rotatedQuadrupole2[4]*Vji[6]
             + rotatedQuadrupole2[1]*Vji[7]
             + rotatedQuadrupole2[2]*Vji[8]
             - sqrtSix*rotatedOctopole2[1]*Vji[9]
             + (sqrtSix*rotatedOctopole2[0]-sqrtFiveHalves*rotatedOctopole2[3])*Vji[10]
             - sqrtFiveHalves*rotatedOctopole2[4]*Vji[11]
             + (sqrtFiveHalves*rotatedOctopole2[1]-sqrtThreeHalves*rotatedOctopole2[5])*Vji[12]
             + (sqrtFiveHalves*rotatedOctopole2[2]-sqrtThreeHalves*rotatedOctopole2[6])*Vji[13]
             + sqrtThreeHalves*rotatedOctopole2[3]*Vji[14]
             + sqrtThreeHalves*rotatedOctopole2[4]*Vji[15]
             ;
    real EJZ = 0
             - rotatedDipole2.z*Vji[2]
             + rotatedDipole2.y*Vji[3]
             - rotatedQuadrupole2[2]*Vji[5]
             + rotatedQuadrupole2[1]*Vji[6]
             - 2*rotatedQuadrupole2[4]*Vji[7]
             + 2*rotatedQuadrupole2[3]*Vji[8]
             - rotatedOctopole2[2]*Vji[10]
             + rotatedOctopole2[1]*Vji[11]
             - 2*rotatedOctopole2[4]*Vji[12]
             + 2*rotatedOctopole2[3]*Vji[13]
             - 3*rotatedOctopole2[6]*Vji[14]
             + 3*rotatedOctopole2[5]*Vji[15]
             ;
    // Define the torque intermediates for the induced dipoles. These are simply the induced dipole torque
    // intermediates dotted with the field due to permanent moments only, at each center. We inline the
    // induced dipole torque intermediates here, for simplicity. N.B. There are no torques on the dipoles
    // themselves, so we accumulate the torque intermediates into separate variables to allow them to be
    // used only in the force calculation.
    //
    // The torque about the x axis (needed to obtain the y force on the induced dipoles, below)
    //    qiUindIx[0] = qiQUindI[2];    qiUindIx[1] = 0;    qiUindIx[2] = -qiQUindI[0]
    real iEIX = qiUindI.z*Vijd[0] - qiUindI.x*Vijd[2];
    real iEJX = qiUindJ.z*Vjid[0] - qiUindJ.x*Vjid[2];
    // The torque about the y axis (needed to obtain the x force on the induced dipoles, below)
    //    qiUindIy[0] = -qiQUindI[1];   qiUindIy[1] = qiQUindI[0];    qiUindIy[2] = 0
    real iEIY = qiUindI.x*Vijd[1] - qiUindI.y*Vijd[0];
    real iEJY = qiUindJ.x*Vjid[1] - qiUindJ.y*Vjid[0];
    // The torque about the z axis (needed to obtain the x force on the induced dipoles, below)
    //    qiUindIz[0] = 0;  qiUindIz[1] = -qiQUindI[2];    qiUindIz[2] = qiQUindI[1]
    real iEIZ = qiUindI.y*Vijd[2] - qiUindI.z*Vijd[1];
    real iEJZ = qiUindJ.y*Vjid[2] - qiUindJ.z*Vjid[1];
#ifdef MUTUAL_POLARIZATION
    // Uind-Uind terms (m=0)
    real eCoef = -2*fourThirds*rInvVec[3]*(3*(thole_d0 + bVec[3]) + alphaRVec[3]*X);
    real dCoef = 2*rInvVec[4]*(6*(dthole_d0 + bVec[3]) + 4*alphaRVec[5]*X);
    iEIX += eCoef*qiUindI.z*qiUindJ.x;
    iEJX += eCoef*qiUindJ.z*qiUindI.x;
    iEIY -= eCoef*qiUindI.y*qiUindJ.x;
    iEJY -= eCoef*qiUindJ.y*qiUindI.x;
    fIZ  += dCoef*qiUindI.x*qiUindJ.x;
    fJZ  += dCoef*qiUindJ.x*qiUindI.x;
    // Uind-Uind terms (m=1)
    eCoef = 4*rInvVec[3]*(thole_d1 + bVec[3] - twoThirds*alphaRVec[3]*X);
    dCoef = -6*rInvVec[4]*(dthole_d1 + bVec[3]);
    iEIX -= eCoef*qiUindI.x*qiUindJ.z;
    iEJX -= eCoef*qiUindJ.x*qiUindI.z;
    iEIY += eCoef*qiUindI.x*qiUindJ.y;
    iEJY += eCoef*qiUindJ.x*qiUindI.y;
    iEIZ += eCoef*qiUindI.y*qiUindJ.z;
    iEJZ += eCoef*qiUindJ.y*qiUindI.z;
    fIZ  += dCoef*(qiUindI.y*qiUindJ.y + qiUindI.z*qiUindJ.z);
    fJZ  += dCoef*(qiUindJ.y*qiUindI.y + qiUindJ.z*qiUindI.z);
#endif

    // The quasi-internal frame forces and torques.  Note that the induced torque intermediates are
    // used in the force expression, but not in the torques; the induced dipoles are isotropic.
    // The quasi-internal frame forces and torques.  Note that the induced torque intermediates are
    // used in the force expression, but not in the torques; the induced dipoles are isotropic.
    real qiForce[3] = {rInv*(EIY+EJY+iEIY+iEJY), -rInv*(EIX+EJX+iEIX+iEJX), -(fJZ+fIZ)};
    real qiTorqueI[3] = {-EIX, -EIY, -EIZ};
    real qiTorqueJ[3] = {-EJX, -EJY, -EJZ};

    if(atom1.damp > 0){
        qiTorqueI[0] += -iEIX;
        qiTorqueI[1] += -iEIY;
        qiTorqueI[2] += -iEIZ;
    }
    if(atom2.damp > 0){
        qiTorqueJ[0] += -iEJX;
        qiTorqueJ[1] += -iEJY;
        qiTorqueJ[2] += -iEJZ;
    }


    real3 force = make_real3(qiRotationMatrix[1][1]*qiForce[0] + qiRotationMatrix[2][1]*qiForce[1] + qiRotationMatrix[0][1]*qiForce[2],
                             qiRotationMatrix[1][2]*qiForce[0] + qiRotationMatrix[2][2]*qiForce[1] + qiRotationMatrix[0][2]*qiForce[2],
                             qiRotationMatrix[1][0]*qiForce[0] + qiRotationMatrix[2][0]*qiForce[1] + qiRotationMatrix[0][0]*qiForce[2]);
    atom1.force += force;
    atom1.torque += make_real3(qiRotationMatrix[1][1]*qiTorqueI[0] + qiRotationMatrix[2][1]*qiTorqueI[1] + qiRotationMatrix[0][1]*qiTorqueI[2],
                               qiRotationMatrix[1][2]*qiTorqueI[0] + qiRotationMatrix[2][2]*qiTorqueI[1] + qiRotationMatrix[0][2]*qiTorqueI[2],
                               qiRotationMatrix[1][0]*qiTorqueI[0] + qiRotationMatrix[2][0]*qiTorqueI[1] + qiRotationMatrix[0][0]*qiTorqueI[2]);
    if (forceFactor == 1) {
        atom2.force -= force;
        atom2.torque += make_real3(qiRotationMatrix[1][1]*qiTorqueJ[0] + qiRotationMatrix[2][1]*qiTorqueJ[1] + qiRotationMatrix[0][1]*qiTorqueJ[2],
                                   qiRotationMatrix[1][2]*qiTorqueJ[0] + qiRotationMatrix[2][2]*qiTorqueJ[1] + qiRotationMatrix[0][2]*qiTorqueJ[2],
                                   qiRotationMatrix[1][0]*qiTorqueJ[0] + qiRotationMatrix[2][0]*qiTorqueJ[1] + qiRotationMatrix[0][0]*qiTorqueJ[2]);
    }
}

/**
 * Compute the self energy and self torque.
 */
__device__ void computeSelfEnergyAndTorque(AtomData& atom1, mixed& energy, const real3& cphi) {
    real cii = atom1.q*atom1.q;
    real3 dipole = make_real3(atom1.sphericalDipole.y, atom1.sphericalDipole.z, atom1.sphericalDipole.x);
    real dii = dot(dipole, dipole+atom1.inducedDipole);
#ifdef INCLUDE_QUADRUPOLES
    real qii = (atom1.sphericalQuadrupole[0]*atom1.sphericalQuadrupole[0] +
                atom1.sphericalQuadrupole[1]*atom1.sphericalQuadrupole[1] +
                atom1.sphericalQuadrupole[2]*atom1.sphericalQuadrupole[2] +
                atom1.sphericalQuadrupole[3]*atom1.sphericalQuadrupole[3] +
                atom1.sphericalQuadrupole[4]*atom1.sphericalQuadrupole[4]);
#else
    real qii = 0;
#endif
#ifdef INCLUDE_OCTOPOLES
    real oii = (atom1.sphericalOctopole[0]*atom1.sphericalOctopole[0] +
                atom1.sphericalOctopole[1]*atom1.sphericalOctopole[1] +
                atom1.sphericalOctopole[2]*atom1.sphericalOctopole[2] +
                atom1.sphericalOctopole[3]*atom1.sphericalOctopole[3] +
                atom1.sphericalOctopole[4]*atom1.sphericalOctopole[4] +
                atom1.sphericalOctopole[5]*atom1.sphericalOctopole[5] +
                atom1.sphericalOctopole[6]*atom1.sphericalOctopole[6]);
#else
    real oii = 0;
#endif
    real prefac = -EWALD_ALPHA/SQRT_PI;
    real a2 = EWALD_ALPHA*EWALD_ALPHA;
    real a4 = a2*a2;
    real a6 = a4*a2;
    energy += prefac*(cii + ((real)2/3)*a2*dii + ((real) 4/15)*a4*qii + ((real) 8/105)*a6*oii);

    // self-torque for PME
    real3 ui = atom1.inducedDipole;
    bool isIsotropic = atom1.damp < 0;
    prefac = isIsotropic ? 4*a2*EWALD_ALPHA/((real) 3*SQRT_PI) : 0;
    atom1.torque += prefac*cross(dipole, ui);
    // The interaction of induced dipoles with rec-space permanent field
    prefac = isIsotropic ? 0 : 1;
    atom1.torque += prefac*cross(cphi, ui);
}

/**
 * Compute electrostatic interactions.
 */
extern "C" __global__ void computeElectrostatics(
        unsigned long long* __restrict__ forceBuffers, unsigned long long* __restrict__ torqueBuffers, mixed* __restrict__ energyBuffer,
        const real4* __restrict__ posq, const uint2* __restrict__ covalentFlags,
        const int2* __restrict__ exclusionTiles, unsigned int startTileIndex, unsigned int numTileIndices,
#ifdef USE_CUTOFF
        const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize,
        real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ, unsigned int maxTiles, const real4* __restrict__ blockCenter,
        const unsigned int* __restrict__ interactingAtoms,
#endif
        const real* __restrict__ sphericalDipole, const real* __restrict__ sphericalQuadrupole, const real* __restrict__ sphericalOctopole,
        const real* __restrict__ inducedDipole, const float2* __restrict__ dampingAndThole, const real* __restrict__ cphi) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    mixed energy = 0;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];

    // First loop: process tiles that contain exclusions.
    
    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const int2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        AtomData data;
        unsigned int atom1 = x*TILE_SIZE + tgx;
        loadAtomData(data, atom1, posq, sphericalDipole, sphericalQuadrupole, sphericalOctopole, inducedDipole, dampingAndThole);
        data.force = make_real3(0);
        data.torque = make_real3(0);
        uint2 covalent = covalentFlags[pos*TILE_SIZE+tgx];
        if (x == y) {
            // This tile is on the diagonal.

            localData[threadIdx.x].pos = data.pos;
            localData[threadIdx.x].q = data.q;
            localData[threadIdx.x].sphericalDipole = data.sphericalDipole;
#ifdef INCLUDE_QUADRUPOLES
            localData[threadIdx.x].sphericalQuadrupole[0] = data.sphericalQuadrupole[0];
            localData[threadIdx.x].sphericalQuadrupole[1] = data.sphericalQuadrupole[1];
            localData[threadIdx.x].sphericalQuadrupole[2] = data.sphericalQuadrupole[2];
            localData[threadIdx.x].sphericalQuadrupole[3] = data.sphericalQuadrupole[3];
            localData[threadIdx.x].sphericalQuadrupole[4] = data.sphericalQuadrupole[4];
#endif
#ifdef INCLUDE_OCTOPOLES
            localData[threadIdx.x].sphericalOctopole[0] = data.sphericalOctopole[0];
            localData[threadIdx.x].sphericalOctopole[1] = data.sphericalOctopole[1];
            localData[threadIdx.x].sphericalOctopole[2] = data.sphericalOctopole[2];
            localData[threadIdx.x].sphericalOctopole[3] = data.sphericalOctopole[3];
            localData[threadIdx.x].sphericalOctopole[4] = data.sphericalOctopole[4];
            localData[threadIdx.x].sphericalOctopole[5] = data.sphericalOctopole[5];
            localData[threadIdx.x].sphericalOctopole[6] = data.sphericalOctopole[6];
#endif
            localData[threadIdx.x].inducedDipole = data.inducedDipole;
            localData[threadIdx.x].thole = data.thole;
            localData[threadIdx.x].damp = data.damp;

            // Compute forces.

            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+j;
                if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    float m = computeMScaleFactor(covalent, j);
                    float p = m;
                    computeOneInteraction(data, localData[tbx+j], true, p, m, 0.5f, energy, periodicBoxSize, invPeriodicBoxSize, periodicBoxVecX, periodicBoxVecY, periodicBoxVecZ);
                }
            }
            if (atom1 < NUM_ATOMS)
                computeSelfEnergyAndTorque(data, energy, make_real3(cphi[20*atom1+1], cphi[20*atom1+2], cphi[20*atom1+3]));
            data.force *= -ENERGY_SCALE_FACTOR;
            data.torque *= ENERGY_SCALE_FACTOR;
            atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            atomicAdd(&torqueBuffers[atom1], static_cast<unsigned long long>((long long) (data.torque.x*0x100000000)));
            atomicAdd(&torqueBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.torque.y*0x100000000)));
            atomicAdd(&torqueBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.torque.z*0x100000000)));
        }
        else {
            // This is an off-diagonal tile.

            unsigned int j = y*TILE_SIZE + tgx;
            loadAtomData(localData[threadIdx.x], j, posq, sphericalDipole, sphericalQuadrupole, sphericalOctopole, inducedDipole, dampingAndThole);
            localData[threadIdx.x].force = make_real3(0);
            localData[threadIdx.x].torque = make_real3(0);
            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    float m = computeMScaleFactor(covalent, tj);
                    float p = m;
                    computeOneInteraction(data, localData[tbx+tj], true, p, m, 1, energy, periodicBoxSize, invPeriodicBoxSize, periodicBoxVecX, periodicBoxVecY, periodicBoxVecZ);
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= -ENERGY_SCALE_FACTOR;
            data.torque *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= -ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].torque *= ENERGY_SCALE_FACTOR;
            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (data.torque.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.torque.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.torque.z*0x100000000)));
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].torque.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].torque.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].torque.z*0x100000000)));
        }
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    if (numTiles > maxTiles)
        return; // There wasn't enough memory for the neighbor list.
    int pos = (int) (numTiles > maxTiles ? startTileIndex+warp*(long long)numTileIndices/totalWarps : warp*(long long)numTiles/totalWarps);
    int end = (int) (numTiles > maxTiles ? startTileIndex+(warp+1)*(long long)numTileIndices/totalWarps : (warp+1)*(long long)numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    int pos = (int) (startTileIndex+warp*(long long)numTiles/totalWarps);
    int end = (int) (startTileIndex+(warp+1)*(long long)numTiles/totalWarps);
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;

    while (pos < end) {
        bool includeTile = true;

        // Extract the coordinates of this tile.

        int x, y;
#ifdef USE_CUTOFF
        x = tiles[pos];
#else
        y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
        x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
            y += (x < y ? -1 : 1);
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        }

        // Skip over tiles that have exclusions, since they were already processed.

        while (skipTiles[tbx+TILE_SIZE-1] < pos) {
            if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                int2 tile = exclusionTiles[skipBase+tgx];
                skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
            }
            else
                skipTiles[threadIdx.x] = end;
            skipBase += TILE_SIZE;            
            currentSkipIndex = tbx;
        }
        while (skipTiles[currentSkipIndex] < pos)
            currentSkipIndex++;
        includeTile = (skipTiles[currentSkipIndex] != pos);
#endif
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.

            AtomData data;
            loadAtomData(data, atom1, posq, sphericalDipole, sphericalQuadrupole, sphericalOctopole, inducedDipole, dampingAndThole);
            data.force = make_real3(0);
            data.torque = make_real3(0);
#ifdef USE_CUTOFF
            unsigned int j = interactingAtoms[pos*TILE_SIZE+tgx];
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
            loadAtomData(localData[threadIdx.x], j, posq, sphericalDipole, sphericalQuadrupole, sphericalOctopole, inducedDipole, dampingAndThole);
            localData[threadIdx.x].force = make_real3(0);
            localData[threadIdx.x].torque = make_real3(0);

            // Compute forces.

            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = atomIndices[tbx+tj];
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    computeOneInteraction(data, localData[tbx+tj], false, 1, 1, 1, energy, periodicBoxSize, invPeriodicBoxSize, periodicBoxVecX, periodicBoxVecY, periodicBoxVecZ);
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= -ENERGY_SCALE_FACTOR;
            data.torque *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= -ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].torque *= ENERGY_SCALE_FACTOR;

            // Write results.

            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (data.torque.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.torque.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.torque.z*0x100000000)));
#ifdef USE_CUTOFF
            offset = atomIndices[threadIdx.x];
#else
            offset = y*TILE_SIZE + tgx;
#endif
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].torque.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].torque.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].torque.z*0x100000000)));
        }
        pos++;
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy*ENERGY_SCALE_FACTOR;
}
