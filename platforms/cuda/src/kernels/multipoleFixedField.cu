#include "hip/hip_runtime.h"
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

typedef struct {
    real4 posq;
    real3 field, dipole;
#ifdef INCLUDE_QUADRUPOLES
    real quadrupoleXX, quadrupoleXY, quadrupoleXZ;
    real quadrupoleYY, quadrupoleYZ, quadrupoleZZ;
#endif
#ifdef INCLUDE_OCTOPOLES
    real octopoleXXX, octopoleXXY, octopoleXXZ, octopoleXYY, octopoleXYZ;
    real octopoleXZZ, octopoleYYY, octopoleYYZ, octopoleYZZ, octopoleZZZ;
#endif
    float thole, damp;
} AtomData;

inline __device__ void loadAtomData(AtomData& data, int atom, const real4* __restrict__ posq, const real* __restrict__ labFrameDipole,
                                   const real* __restrict__ labFrameQuadrupole, const real* __restrict__ labFrameOctopole,
                                   const float2* __restrict__ dampingAndThole) {
    data.posq = posq[atom];
    data.dipole.x = labFrameDipole[atom*3];
    data.dipole.y = labFrameDipole[atom*3+1];
    data.dipole.z = labFrameDipole[atom*3+2];
#ifdef INCLUDE_QUADRUPOLES
    data.quadrupoleXX = labFrameQuadrupole[atom*5];
    data.quadrupoleXY = labFrameQuadrupole[atom*5+1];
    data.quadrupoleXZ = labFrameQuadrupole[atom*5+2];
    data.quadrupoleYY = labFrameQuadrupole[atom*5+3];
    data.quadrupoleYZ = labFrameQuadrupole[atom*5+4];
    data.quadrupoleZZ = -(data.quadrupoleXX+data.quadrupoleYY);
#endif
#ifdef INCLUDE_OCTOPOLES
    data.octopoleXXX = labFrameOctopole[atom*7+0];
    data.octopoleXXY = labFrameOctopole[atom*7+1];
    data.octopoleXXZ = labFrameOctopole[atom*7+2];
    data.octopoleXYY = labFrameOctopole[atom*7+3];
    data.octopoleXYZ = labFrameOctopole[atom*7+4];
    data.octopoleYYY = labFrameOctopole[atom*7+5];
    data.octopoleYYZ = labFrameOctopole[atom*7+6];
    data.octopoleXZZ = -data.octopoleXXX-data.octopoleXYY;
    data.octopoleYZZ = -data.octopoleXXY-data.octopoleYYY;
    data.octopoleZZZ = -data.octopoleXXZ-data.octopoleYYZ;
#endif
    float2 temp = dampingAndThole[atom];
    data.damp = temp.x;
    data.thole = temp.y;
}

#ifdef USE_EWALD
__device__ void computeOneInteraction(AtomData& atom1, AtomData& atom2, real3 deltaR, float pScale, real3* fields) {
    real r2 = dot(deltaR, deltaR);
    if (r2 <= CUTOFF_SQUARED) {
        // calculate the error function damping terms

        real r = SQRT(r2);
        real ralpha = EWALD_ALPHA*r;
        real exp2a = EXP(-(ralpha*ralpha));
#ifdef USE_DOUBLE_PRECISION
        const real erfcAlphaR = erfc(ralpha);
#else
        // This approximation for erfc is from Abramowitz and Stegun (1964) p. 299.  They cite the following as
        // the original source: C. Hastings, Jr., Approximations for Digital Computers (1955).  It has a maximum
        // error of 1.5e-7.

        const real t = RECIP(1.0f+0.3275911f*ralpha);
        const real erfcAlphaR = (0.254829592f+(-0.284496736f+(1.421413741f+(-1.453152027f+1.061405429f*t)*t)*t)*t)*t*exp2a;
#endif
        real bn0 = erfcAlphaR/r;
        real alsq2 = 2*EWALD_ALPHA*EWALD_ALPHA;
        real alsq2n = RECIP(SQRT_PI*EWALD_ALPHA);
        alsq2n *= alsq2;
        real bn1 = (bn0+alsq2n*exp2a)/r2;
        alsq2n *= alsq2;
        real bn2 = (3*bn1+alsq2n*exp2a)/r2;

        // compute the error function scaled and unscaled terms

        real damp = fabs(atom1.damp*atom2.damp);
        real pgamma = (pScale == 0 ? atom1.thole + atom2.thole : DEFAULT_THOLE_WIDTH);
        real dfac = (damp == 0 ? 9999 : pgamma * r / damp); // TODO the inverses should be computed at parse time
        real expdamp = (dfac < 50 ? EXP(-dfac) : 0);

        real scale3 = 1 - expdamp*(1 + dfac + 0.5f*dfac*dfac);
        real scale5 = 1 - expdamp*(1 + dfac + 0.5f*dfac*dfac + dfac*dfac*dfac/6);

        real psc3 = pScale*scale3;
        real psc5 = pScale*scale5;

        real r3 = r*r2;
        real r5 = r3*r2;

        real prr3 = (1-psc3)/r3;
        real prr5 = 3*(1-psc5)/r5;

        real dir = dot(atom1.dipole, deltaR);
        real dkr = dot(atom2.dipole, deltaR);

#ifdef INCLUDE_QUADRUPOLES
        alsq2n *= alsq2;
        real bn3 = (5*bn2+alsq2n*exp2a)/r2;
        real scale7 = 1 - expdamp*(1 + dfac + 0.5f*dfac*dfac + dfac*dfac*dfac/6 + dfac*dfac*dfac*dfac/30);
        real psc7 = pScale*scale7;
        real r7 = r5*r2;
        real prr7 = 15*(1-psc7)/r7;

        real3 qi;
        qi.x = atom1.quadrupoleXX*deltaR.x + atom1.quadrupoleXY*deltaR.y + atom1.quadrupoleXZ*deltaR.z;
        qi.y = atom1.quadrupoleXY*deltaR.x + atom1.quadrupoleYY*deltaR.y + atom1.quadrupoleYZ*deltaR.z;
        qi.z = atom1.quadrupoleXZ*deltaR.x + atom1.quadrupoleYZ*deltaR.y + atom1.quadrupoleZZ*deltaR.z;
        real qir = dot(qi, deltaR);

        real3 qk;
        qk.x = atom2.quadrupoleXX*deltaR.x + atom2.quadrupoleXY*deltaR.y + atom2.quadrupoleXZ*deltaR.z;
        qk.y = atom2.quadrupoleXY*deltaR.x + atom2.quadrupoleYY*deltaR.y + atom2.quadrupoleYZ*deltaR.z;
        qk.z = atom2.quadrupoleXZ*deltaR.x + atom2.quadrupoleYZ*deltaR.y + atom2.quadrupoleZZ*deltaR.z;
        real qkr = dot(qk, deltaR);

        real3 fim = -deltaR*(bn1*atom2.posq.w-bn2*dkr+bn3*qkr) - bn1*atom2.dipole + 2*bn2*qk;
        real3 fkm = deltaR*(bn1*atom1.posq.w+bn2*dir+bn3*qir) - bn1*atom1.dipole - 2*bn2*qi;
        real3 fip = -deltaR*(prr3*atom2.posq.w-prr5*dkr+prr7*qkr) - prr3*atom2.dipole + 2*prr5*qk;
        real3 fkp = deltaR*(prr3*atom1.posq.w+prr5*dir+prr7*qir) - prr3*atom1.dipole - 2*prr5*qi;
#ifdef INCLUDE_OCTOPOLES
        alsq2n *= alsq2;
        real bn4 = (7*bn3+alsq2n*exp2a)/r2;
        real scale9 = 1 - expdamp*(1 + dfac + 0.5f*dfac*dfac + dfac*dfac*dfac/6 + 4*dfac*dfac*dfac*dfac/105 + dfac*dfac*dfac*dfac*dfac/210);
        real psc9 = pScale*scale9;
        real r9 = r7*r2;
        real prr9 = 105*(1-psc9)/r9;

        real3 oxx = make_real3(atom2.octopoleXXX, atom2.octopoleXXY, atom2.octopoleXXZ);
        real3 oxy = make_real3(atom2.octopoleXXY, atom2.octopoleXYY, atom2.octopoleXYZ);
        real3 oxz = make_real3(atom2.octopoleXXZ, atom2.octopoleXYZ, atom2.octopoleXZZ);
        real3 oyy = make_real3(atom2.octopoleXYY, atom2.octopoleYYY, atom2.octopoleYYZ);
        real3 oyz = make_real3(atom2.octopoleXYZ, atom2.octopoleYYZ, atom2.octopoleYZZ);
        real3 ozz = make_real3(atom2.octopoleXZZ, atom2.octopoleYZZ, atom2.octopoleZZZ);
        real3 ox  = make_real3(dot(oxx,deltaR), dot(oxy,deltaR), dot(oxz,deltaR));
        real3 oy  = make_real3(dot(oxy,deltaR), dot(oyy,deltaR), dot(oyz,deltaR));
        real3 oz  = make_real3(dot(oxz,deltaR), dot(oyz,deltaR), dot(ozz,deltaR));
        real3 o   = make_real3(dot(ox,deltaR), dot(oy,deltaR), dot(oz,deltaR));
        fim += -o*(3*bn3) + deltaR*bn4*dot(o,deltaR);
        fip += -o*(3*prr7) + deltaR*prr9*dot(o,deltaR);

        oxx = make_real3(atom1.octopoleXXX, atom1.octopoleXXY, atom1.octopoleXXZ);
        oxy = make_real3(atom1.octopoleXXY, atom1.octopoleXYY, atom1.octopoleXYZ);
        oxz = make_real3(atom1.octopoleXXZ, atom1.octopoleXYZ, atom1.octopoleXZZ);
        oyy = make_real3(atom1.octopoleXYY, atom1.octopoleYYY, atom1.octopoleYYZ);
        oyz = make_real3(atom1.octopoleXYZ, atom1.octopoleYYZ, atom1.octopoleYZZ);
        ozz = make_real3(atom1.octopoleXZZ, atom1.octopoleYZZ, atom1.octopoleZZZ);
        ox  = make_real3(dot(oxx,deltaR), dot(oxy,deltaR), dot(oxz,deltaR));
        oy  = make_real3(dot(oxy,deltaR), dot(oyy,deltaR), dot(oyz,deltaR));
        oz  = make_real3(dot(oxz,deltaR), dot(oyz,deltaR), dot(ozz,deltaR));
        o   = make_real3(dot(ox,deltaR), dot(oy,deltaR), dot(oz,deltaR));
        fkm += -o*(3*bn3) + deltaR*bn4*dot(o,deltaR);
        fkp += -o*(3*prr7) + deltaR*prr9*dot(o,deltaR);
#endif // End INCLUDE_OCTOPOLES
#else
        // Charge-only routine
        real3 fim = -deltaR*(bn1*atom2.posq.w-bn2*dkr) - bn1*atom2.dipole;
        real3 fkm = deltaR*(bn1*atom1.posq.w+bn2*dir) - bn1*atom1.dipole;
        real3 fip = -deltaR*(prr3*atom2.posq.w-prr5*dkr) - prr3*atom2.dipole;
        real3 fkp = deltaR*(prr3*atom1.posq.w+prr5*dir) - prr3*atom1.dipole;
#endif
        // increment the field at each site due to this interaction
        fields[0] = fim-fip;
        fields[1] = fkm-fkp;
    }
    else {
        fields[0] = make_real3(0);
        fields[1] = make_real3(0);
    }
}
#else
__device__ void computeOneInteraction(AtomData& atom1, AtomData& atom2, real3 deltaR, float pScale, real3* fields) {
    real rI = RSQRT(dot(deltaR, deltaR));
    real r = RECIP(rI);
    real r2I = rI*rI;

    real rr3 = rI*r2I;
    real rr5 = 3*rr3*r2I;
    real rr7 = 5*rr5*r2I;
    real rr9 = 7*rr7*r2I;
 
    // get scaling factors, if needed
    
    float damp = fabs(atom1.damp*atom2.damp);
    real dampExp;
    if (damp != 0) {

        // get scaling factors
      
        real ratio = r/damp;
        float pGamma  = pScale == 0.0f ? atom1.thole + atom2.thole : DEFAULT_THOLE_WIDTH;
        damp = ratio*pGamma;
        dampExp = EXP(-damp);
    }
    else
        dampExp = 0;
    rr3 *= 1 - dampExp*(1 + damp + damp*damp/2);
    rr5 *= 1 - dampExp*(1 + damp + damp*damp/2 + damp*damp*damp/6);
    rr7 *= 1 - dampExp*(1 + damp + damp*damp/2 + damp*damp*damp/6 + damp*damp*damp*damp/30);
    rr9 *= 1 - dampExp*(1 + damp + damp*damp/2 + damp*damp*damp/6 + 4*damp*damp*damp*damp/105 + damp*damp*damp*damp*damp/210);
    real rr5_2 = 2*rr5;
    real rr7_3 = 3*rr7;
 
    real dir = dot(atom1.dipole, deltaR);
    real dkr = dot(atom2.dipole, deltaR);

#ifdef INCLUDE_QUADRUPOLES
    real3 qi;
    qi.x = atom1.quadrupoleXX*deltaR.x + atom1.quadrupoleXY*deltaR.y + atom1.quadrupoleXZ*deltaR.z;
    qi.y = atom1.quadrupoleXY*deltaR.x + atom1.quadrupoleYY*deltaR.y + atom1.quadrupoleYZ*deltaR.z;
    qi.z = atom1.quadrupoleXZ*deltaR.x + atom1.quadrupoleYZ*deltaR.y + atom1.quadrupoleZZ*deltaR.z;
    real qir = dot(qi, deltaR);

    real3 qk;
    qk.x = atom2.quadrupoleXX*deltaR.x + atom2.quadrupoleXY*deltaR.y + atom2.quadrupoleXZ*deltaR.z;
    qk.y = atom2.quadrupoleXY*deltaR.x + atom2.quadrupoleYY*deltaR.y + atom2.quadrupoleYZ*deltaR.z;
    qk.z = atom2.quadrupoleXZ*deltaR.x + atom2.quadrupoleYZ*deltaR.y + atom2.quadrupoleZZ*deltaR.z;
    real qkr = dot(qk, deltaR);

    real factor = -rr3*atom2.posq.w + rr5*dkr - rr7*qkr;
    real3 field1 = deltaR*factor - rr3*atom2.dipole + rr5_2*qk;
    factor = rr3*atom1.posq.w + rr5*dir + rr7*qir;
    real3 field2 = deltaR*factor - rr3*atom1.dipole - rr5_2*qi;
#ifdef INCLUDE_OCTOPOLES
    real3 oxx = make_real3(atom2.octopoleXXX, atom2.octopoleXXY, atom2.octopoleXXZ);
    real3 oxy = make_real3(atom2.octopoleXXY, atom2.octopoleXYY, atom2.octopoleXYZ);
    real3 oxz = make_real3(atom2.octopoleXXZ, atom2.octopoleXYZ, atom2.octopoleXZZ);
    real3 oyy = make_real3(atom2.octopoleXYY, atom2.octopoleYYY, atom2.octopoleYYZ);
    real3 oyz = make_real3(atom2.octopoleXYZ, atom2.octopoleYYZ, atom2.octopoleYZZ);
    real3 ozz = make_real3(atom2.octopoleXZZ, atom2.octopoleYZZ, atom2.octopoleZZZ);
    real3 ox  = make_real3(dot(oxx,deltaR), dot(oxy,deltaR), dot(oxz,deltaR));
    real3 oy  = make_real3(dot(oxy,deltaR), dot(oyy,deltaR), dot(oyz,deltaR));
    real3 oz  = make_real3(dot(oxz,deltaR), dot(oyz,deltaR), dot(ozz,deltaR));
    real3 o   = make_real3(dot(ox,deltaR), dot(oy,deltaR), dot(oz,deltaR));
    field1 += deltaR*rr9*dot(o,deltaR) - o*rr7_3;
    oxx = make_real3(atom1.octopoleXXX, atom1.octopoleXXY, atom1.octopoleXXZ);
    oxy = make_real3(atom1.octopoleXXY, atom1.octopoleXYY, atom1.octopoleXYZ);
    oxz = make_real3(atom1.octopoleXXZ, atom1.octopoleXYZ, atom1.octopoleXZZ);
    oyy = make_real3(atom1.octopoleXYY, atom1.octopoleYYY, atom1.octopoleYYZ);
    oyz = make_real3(atom1.octopoleXYZ, atom1.octopoleYYZ, atom1.octopoleYZZ);
    ozz = make_real3(atom1.octopoleXZZ, atom1.octopoleYZZ, atom1.octopoleZZZ);
    ox  = make_real3(dot(oxx,deltaR), dot(oxy,deltaR), dot(oxz,deltaR));
    oy  = make_real3(dot(oxy,deltaR), dot(oyy,deltaR), dot(oyz,deltaR));
    oz  = make_real3(dot(oxz,deltaR), dot(oyz,deltaR), dot(ozz,deltaR));
    o   = make_real3(dot(ox,deltaR), dot(oy,deltaR), dot(oz,deltaR));
    field2 += deltaR*rr9*dot(o,deltaR) - o*rr7_3;
#endif
#else
    real factor = -rr3*atom2.posq.w + rr5*dkr;
    real3 field1 = deltaR*factor - rr3*atom2.dipole;
    factor = rr3*atom1.posq.w + rr5*dir;
    real3 field2 = deltaR*factor - rr3*atom1.dipole;
#endif
    fields[0] = pScale*field1;
    fields[1] = pScale*field2;
}
#endif


__device__ float computePScaleFactor(uint2 covalent, int index) {
    int mask = 1<<index;
    bool x = (covalent.x & mask);
    bool y = (covalent.y & mask);
	return (x ? (y ? 0.0f : (float) SCALEFACTOR14): 1.0f);
}

/**
 * Compute nonbonded interactions.
 */
extern "C" __global__ void computeFixedField(
        unsigned long long* __restrict__ fieldBuffers, const real4* __restrict__ posq,
        const uint2* __restrict__ covalentFlags, const int2* __restrict__ exclusionTiles,
        unsigned int startTileIndex, unsigned int numTileIndices,
#ifdef USE_CUTOFF
        const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize,
        real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ, unsigned int maxTiles, const real4* __restrict__ blockCenter,
        const unsigned int* __restrict__ interactingAtoms,
#endif
        const real* __restrict__ labFrameDipole, const real* __restrict__ labFrameQuadrupole, const real* __restrict__ labFrameOctopole,
        const float2* __restrict__ dampingAndThole) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];

    // First loop: process tiles that contain exclusions.

    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const int2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        AtomData data;
        data.field = make_real3(0);
        unsigned int atom1 = x*TILE_SIZE + tgx;
        loadAtomData(data, atom1, posq, labFrameDipole, labFrameQuadrupole, labFrameOctopole, dampingAndThole);
        uint2 covalent = covalentFlags[pos*TILE_SIZE+tgx];
        if (x == y) {
            // This tile is on the diagonal.

            const unsigned int localAtomIndex = threadIdx.x;
            localData[localAtomIndex].posq = data.posq;
            localData[localAtomIndex].dipole = data.dipole;
#ifdef INCLUDE_QUADRUPOLES
            localData[localAtomIndex].quadrupoleXX = data.quadrupoleXX;
            localData[localAtomIndex].quadrupoleXY = data.quadrupoleXY;
            localData[localAtomIndex].quadrupoleXZ = data.quadrupoleXZ;
            localData[localAtomIndex].quadrupoleYY = data.quadrupoleYY;
            localData[localAtomIndex].quadrupoleYZ = data.quadrupoleYZ;
            localData[localAtomIndex].quadrupoleZZ = data.quadrupoleZZ;
#endif
#ifdef INCLUDE_OCTOPOLES
            localData[localAtomIndex].octopoleXXX = data.octopoleXXX;
            localData[localAtomIndex].octopoleXXY = data.octopoleXXY;
            localData[localAtomIndex].octopoleXXZ = data.octopoleXXZ;
            localData[localAtomIndex].octopoleXYY = data.octopoleXYY;
            localData[localAtomIndex].octopoleXYZ = data.octopoleXYZ;
            localData[localAtomIndex].octopoleXZZ = data.octopoleXZZ;
            localData[localAtomIndex].octopoleYYY = data.octopoleYYY;
            localData[localAtomIndex].octopoleYYZ = data.octopoleYYZ;
            localData[localAtomIndex].octopoleYZZ = data.octopoleYZZ;
            localData[localAtomIndex].octopoleZZZ = data.octopoleZZZ;
#endif
            localData[localAtomIndex].thole = data.thole;
            localData[localAtomIndex].damp = data.damp;
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                real3 delta = trimTo3(localData[tbx+j].posq-data.posq);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                int atom2 = y*TILE_SIZE+j;
                if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 fields[2];
                    float p = computePScaleFactor(covalent, j);
                    computeOneInteraction(data, localData[tbx+j], delta, p, fields);
                    data.field += fields[0];
                }
            }
        }
        else {
            // This is an off-diagonal tile.

            const unsigned int localAtomIndex = threadIdx.x;
            unsigned int j = y*TILE_SIZE + tgx;
            loadAtomData(localData[localAtomIndex], j, posq, labFrameDipole, labFrameQuadrupole, labFrameOctopole, dampingAndThole);
            localData[localAtomIndex].field = make_real3(0);
            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                real3 delta = trimTo3(localData[tbx+tj].posq-data.posq);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 fields[2];
                    float p = computePScaleFactor(covalent, tj);
                    computeOneInteraction(data, localData[tbx+tj], delta, p, fields);
                    data.field += fields[0];
                    localData[tbx+tj].field += fields[1];
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
        }
        
        // Write results.
        
        unsigned int offset = x*TILE_SIZE + tgx;
        atomicAdd(&fieldBuffers[offset], static_cast<unsigned long long>((long long) (data.field.x*0x100000000)));
        atomicAdd(&fieldBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.y*0x100000000)));
        atomicAdd(&fieldBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.z*0x100000000)));
        if (x != y) {
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&fieldBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.x*0x100000000)));
            atomicAdd(&fieldBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.y*0x100000000)));
            atomicAdd(&fieldBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.z*0x100000000)));
        }
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    if (numTiles > maxTiles)
        return; // There wasn't enough memory for the neighbor list.
    int pos = (int) (numTiles > maxTiles ? startTileIndex+warp*(long long)numTileIndices/totalWarps : warp*(long long)numTiles/totalWarps);
    int end = (int) (numTiles > maxTiles ? startTileIndex+(warp+1)*(long long)numTileIndices/totalWarps : (warp+1)*(long long)numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    int pos = (int) (startTileIndex+warp*(long long)numTiles/totalWarps);
    int end = (int) (startTileIndex+(warp+1)*(long long)numTiles/totalWarps);
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;

    while (pos < end) {
        bool includeTile = true;

        // Extract the coordinates of this tile.
        
        int x;
#ifdef USE_CUTOFF
        x = tiles[pos];
#else
        int y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
        x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
            y += (x < y ? -1 : 1);
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        }

        // Skip over tiles that have exclusions, since they were already processed.

        while (skipTiles[tbx+TILE_SIZE-1] < pos) {
            if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                int2 tile = exclusionTiles[skipBase+tgx];
                skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
            }
            else
                skipTiles[threadIdx.x] = end;
            skipBase += TILE_SIZE;            
            currentSkipIndex = tbx;
        }
        while (skipTiles[currentSkipIndex] < pos)
            currentSkipIndex++;
        includeTile = (skipTiles[currentSkipIndex] != pos);
#endif
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.

            AtomData data;
            data.field = make_real3(0);
            loadAtomData(data, atom1, posq, labFrameDipole, labFrameQuadrupole, labFrameOctopole, dampingAndThole);
#ifdef USE_CUTOFF
            unsigned int j = interactingAtoms[pos*TILE_SIZE+tgx];
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
            const unsigned int localAtomIndex = threadIdx.x;
            loadAtomData(localData[localAtomIndex], j, posq, labFrameDipole, labFrameQuadrupole, labFrameOctopole, dampingAndThole);
            localData[localAtomIndex].field = make_real3(0);

            // Compute the full set of interactions in this tile.

            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                real3 delta = trimTo3(localData[tbx+tj].posq-data.posq);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                int atom2 = atomIndices[tbx+tj];
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 fields[2];
                    computeOneInteraction(data, localData[tbx+tj], delta, 1, fields);
                    data.field += fields[0];
                    localData[tbx+tj].field += fields[1];
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }

            // Write results.

            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&fieldBuffers[offset], static_cast<unsigned long long>((long long) (data.field.x*0x100000000)));
            atomicAdd(&fieldBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.y*0x100000000)));
            atomicAdd(&fieldBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.z*0x100000000)));
#ifdef USE_CUTOFF
            offset = atomIndices[threadIdx.x];
#else
            offset = y*TILE_SIZE + tgx;
#endif
            atomicAdd(&fieldBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.x*0x100000000)));
            atomicAdd(&fieldBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.y*0x100000000)));
            atomicAdd(&fieldBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.z*0x100000000)));
        }
        pos++;
    }
}
