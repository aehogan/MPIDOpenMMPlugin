#include "hip/hip_runtime.h"
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

typedef struct {
    real3 pos;
    real3 field, inducedDipole;
#ifdef EXTRAPOLATED_POLARIZATION
    real fieldGradient[6];
#endif
    float thole, damp;
} AtomData;

inline __device__ void loadAtomData(AtomData& data, int atom, const real4* __restrict__ posq, const real* __restrict__ inducedDipole,
        const float2* __restrict__ dampingAndThole) {
    real4 atomPosq = posq[atom];
    data.pos = make_real3(atomPosq.x, atomPosq.y, atomPosq.z);
    data.inducedDipole.x = inducedDipole[atom*3];
    data.inducedDipole.y = inducedDipole[atom*3+1];
    data.inducedDipole.z = inducedDipole[atom*3+2];
    float2 temp = dampingAndThole[atom];
    data.damp = fabs(temp.x);
    data.thole = temp.y;
}

__device__ float computePScaleFactor(uint2 covalent, int index) {
    int mask = 1<<index;
    bool x = (covalent.x & mask);
    bool y = (covalent.y & mask);
	return (x ? (y ? 0.0f : (float) SCALEFACTOR14): 1.0f);
}

inline __device__ void zeroAtomData(AtomData& data) {
    data.field = make_real3(0);
#ifdef EXTRAPOLATED_POLARIZATION
    for (int i = 0; i < 6; i++) {
        data.fieldGradient[i] = 0;
    }
#endif
}

#ifdef EXTRAPOLATED_POLARIZATION
    #define SAVE_ATOM_DATA(index, data) saveAtomData(index, data, field, fieldGradient);
#else
    #define SAVE_ATOM_DATA(index, data) saveAtomData(index, data, field);
#endif

inline __device__ void saveAtomData(int index, AtomData& data, unsigned long long* __restrict__ field
#ifdef EXTRAPOLATED_POLARIZATION
        , unsigned long long* __restrict__ fieldGradient
#endif
        ) {
    atomicAdd(&field[index], static_cast<unsigned long long>((long long) (data.field.x*0x100000000)));
    atomicAdd(&field[index+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.y*0x100000000)));
    atomicAdd(&field[index+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.z*0x100000000)));
#ifdef EXTRAPOLATED_POLARIZATION
    for (int i = 0; i < 6; i++) {
        atomicAdd(&fieldGradient[6*index+i], static_cast<unsigned long long>((long long) (data.fieldGradient[i]*0x100000000)));
    }
#endif
}

#ifdef USE_EWALD
__device__ void computeOneInteraction(AtomData& atom1, AtomData& atom2, real3 deltaR, float pScale, bool isSelfInteraction) {
    if (isSelfInteraction)
        return;
    real scale1, scale2, scale3;
    real r2 = dot(deltaR, deltaR);
    if (r2 < CUTOFF_SQUARED) {
        real rI = RSQRT(r2);
        real r = RECIP(rI);
        real rI2 = rI*rI;

        // calculate the error function damping terms

        real ralpha = EWALD_ALPHA*r;
        real exp2a = EXP(-(ralpha*ralpha));
#ifdef USE_DOUBLE_PRECISION
        const real erfcAlphaR = erfc(ralpha);
#else
        // This approximation for erfc is from Abramowitz and Stegun (1964) p. 299.  They cite the following as
        // the original source: C. Hastings, Jr., Approximations for Digital Computers (1955).  It has a maximum
        // error of 1.5e-7.

        const real t = RECIP(1.0f+0.3275911f*ralpha);
        const real erfcAlphaR = (0.254829592f+(-0.284496736f+(1.421413741f+(-1.453152027f+1.061405429f*t)*t)*t)*t)*t*exp2a;
#endif
        real bn0 = erfcAlphaR*rI;
        real alsq2 = 2*EWALD_ALPHA*EWALD_ALPHA;
        real alsq2n = RECIP(SQRT_PI*EWALD_ALPHA);
        alsq2n *= alsq2;
        real bn1 = (bn0+alsq2n*exp2a)*rI2;

        alsq2n *= alsq2;
        real bn2 = (3*bn1+alsq2n*exp2a)*rI2;

        alsq2n *= alsq2;
        real bn3 = (5*bn2+alsq2n*exp2a)*rI2;

        // compute the error function scaled and unscaled terms

        real damp = fabs(atom1.damp*atom2.damp);
        real ratio = damp == 0 ? 0 : (r/damp);
        float pgamma = pScale == 0 ? atom1.thole + atom2.thole : DEFAULT_THOLE_WIDTH;
        damp = pgamma*ratio;
        real expdamp = ratio == 0 ? 0 : EXP(-damp);
        real dsc3 = 1 - expdamp*(1 + damp + damp*damp/2);
        real dsc5 = 1 - expdamp*(1 + damp + damp*damp/2 + damp*damp*damp/6);
        real dsc7 = 1 - expdamp*(1 + damp + damp*damp/2 + damp*damp*damp/6 + damp*damp*damp*damp/30);
        real r3 = (r*r2);
        real r5 = (r3*r2);
        real r7 = (r5*r2);
        real rr3 = (1-dsc3)/r3;
        real rr5 = 3*(1-dsc5)/r5;
        real rr7 = 15*(1-dsc7)/r7;

        scale1 = rr3 - bn1;
        scale2 = bn2 - rr5;
        scale3 = bn3 - rr7;
    }
    else {
        scale1 = 0;
        scale2 = 0;
        scale3 = 0;
    }
    real dDotDelta = scale2*dot(deltaR, atom2.inducedDipole);
    atom1.field += scale1*atom2.inducedDipole + dDotDelta*deltaR;
    dDotDelta = scale2*dot(deltaR, atom1.inducedDipole);
    atom2.field += scale1*atom1.inducedDipole + dDotDelta*deltaR;
#ifdef EXTRAPOLATED_POLARIZATION
    // Compute and store the field gradients for later use.
    
    real3 dipole = atom1.inducedDipole;
    real muDotR = dipole.x*deltaR.x + dipole.y*deltaR.y + dipole.z*deltaR.z;
    atom2.fieldGradient[0] -= muDotR*deltaR.x*deltaR.x*scale3 - (2*dipole.x*deltaR.x + muDotR)*scale2;
    atom2.fieldGradient[1] -= muDotR*deltaR.y*deltaR.y*scale3 - (2*dipole.y*deltaR.y + muDotR)*scale2;
    atom2.fieldGradient[2] -= muDotR*deltaR.z*deltaR.z*scale3 - (2*dipole.z*deltaR.z + muDotR)*scale2;
    atom2.fieldGradient[3] -= muDotR*deltaR.x*deltaR.y*scale3 - (dipole.x*deltaR.y + dipole.y*deltaR.x)*scale2;
    atom2.fieldGradient[4] -= muDotR*deltaR.x*deltaR.z*scale3 - (dipole.x*deltaR.z + dipole.z*deltaR.x)*scale2;
    atom2.fieldGradient[5] -= muDotR*deltaR.y*deltaR.z*scale3 - (dipole.y*deltaR.z + dipole.z*deltaR.y)*scale2;

    dipole = atom2.inducedDipole;
    muDotR = dipole.x*deltaR.x + dipole.y*deltaR.y + dipole.z*deltaR.z;
    atom1.fieldGradient[0] += muDotR*deltaR.x*deltaR.x*scale3 - (2*dipole.x*deltaR.x + muDotR)*scale2;
    atom1.fieldGradient[1] += muDotR*deltaR.y*deltaR.y*scale3 - (2*dipole.y*deltaR.y + muDotR)*scale2;
    atom1.fieldGradient[2] += muDotR*deltaR.z*deltaR.z*scale3 - (2*dipole.z*deltaR.z + muDotR)*scale2;
    atom1.fieldGradient[3] += muDotR*deltaR.x*deltaR.y*scale3 - (dipole.x*deltaR.y + dipole.y*deltaR.x)*scale2;
    atom1.fieldGradient[4] += muDotR*deltaR.x*deltaR.z*scale3 - (dipole.x*deltaR.z + dipole.z*deltaR.x)*scale2;
    atom1.fieldGradient[5] += muDotR*deltaR.y*deltaR.z*scale3 - (dipole.y*deltaR.z + dipole.z*deltaR.y)*scale2;
#endif
}

#else

__device__ void computeOneInteraction(AtomData& atom1, AtomData& atom2, real3 deltaR, float pScale, bool isSelfInteraction) {
    if (isSelfInteraction)
        return;
    real rI = RSQRT(dot(deltaR, deltaR));
    real r = RECIP(rI);
    real r2I = rI*rI;
    real rr3 = -rI*r2I;
    real rr5 = -3*rr3*r2I;
    real rr7 = 5*rr5*r2I;
    real dampProd = fabs(atom1.damp*atom2.damp);
    real ratio = (dampProd != 0 ? r/dampProd : 1);
    float pGamma  = pScale == 0.0 ? atom1.thole + atom2.thole : DEFAULT_THOLE_WIDTH;
    real damp = ratio*pGamma;
    real dampExp = (dampProd != 0 ? EXP(-damp) : 0); 
    rr3 *= 1 - dampExp*(1 + damp + damp*damp/2);
    rr5 *= 1 - dampExp*(1 + damp + damp*damp/2 + damp*damp*damp/6);
    rr7 *= 1 - dampExp*(1 + damp + damp*damp/2 + damp*damp*damp/6 + damp*damp*damp*damp/30);
    real dDotDelta = rr5*dot(deltaR, atom2.inducedDipole);
    atom1.field += rr3*atom2.inducedDipole + dDotDelta*deltaR;
    dDotDelta = rr5*dot(deltaR, atom1.inducedDipole);
    atom2.field += rr3*atom1.inducedDipole + dDotDelta*deltaR;
#ifdef EXTRAPOLATED_POLARIZATION
    // Compute and store the field gradients for later use.
    real3 dipole = atom1.inducedDipole;
    real muDotR = dipole.x*deltaR.x + dipole.y*deltaR.y + dipole.z*deltaR.z;
    atom2.fieldGradient[0] -= muDotR*deltaR.x*deltaR.x*rr7 - (2*dipole.x*deltaR.x + muDotR)*rr5;
    atom2.fieldGradient[1] -= muDotR*deltaR.y*deltaR.y*rr7 - (2*dipole.y*deltaR.y + muDotR)*rr5;
    atom2.fieldGradient[2] -= muDotR*deltaR.z*deltaR.z*rr7 - (2*dipole.z*deltaR.z + muDotR)*rr5;
    atom2.fieldGradient[3] -= muDotR*deltaR.x*deltaR.y*rr7 - (dipole.x*deltaR.y + dipole.y*deltaR.x)*rr5;
    atom2.fieldGradient[4] -= muDotR*deltaR.x*deltaR.z*rr7 - (dipole.x*deltaR.z + dipole.z*deltaR.x)*rr5;
    atom2.fieldGradient[5] -= muDotR*deltaR.y*deltaR.z*rr7 - (dipole.y*deltaR.z + dipole.z*deltaR.y)*rr5;

    dipole = atom2.inducedDipole;
    muDotR = dipole.x*deltaR.x + dipole.y*deltaR.y + dipole.z*deltaR.z;
    atom1.fieldGradient[0] += muDotR*deltaR.x*deltaR.x*rr7 - (2*dipole.x*deltaR.x + muDotR)*rr5;
    atom1.fieldGradient[1] += muDotR*deltaR.y*deltaR.y*rr7 - (2*dipole.y*deltaR.y + muDotR)*rr5;
    atom1.fieldGradient[2] += muDotR*deltaR.z*deltaR.z*rr7 - (2*dipole.z*deltaR.z + muDotR)*rr5;
    atom1.fieldGradient[3] += muDotR*deltaR.x*deltaR.y*rr7 - (dipole.x*deltaR.y + dipole.y*deltaR.x)*rr5;
    atom1.fieldGradient[4] += muDotR*deltaR.x*deltaR.z*rr7 - (dipole.x*deltaR.z + dipole.z*deltaR.x)*rr5;
    atom1.fieldGradient[5] += muDotR*deltaR.y*deltaR.z*rr7 - (dipole.y*deltaR.z + dipole.z*deltaR.y)*rr5;
#endif
}
#endif

/**
 * Compute the mutual induced field.
 */
extern "C" __global__ void computeInducedField(
        unsigned long long* __restrict__ field, const real4* __restrict__ posq, const uint2* __restrict__ covalentFlags, const int2* __restrict__ exclusionTiles, 
        const real* __restrict__ inducedDipole, unsigned int startTileIndex, unsigned int numTileIndices,
#ifdef EXTRAPOLATED_POLARIZATION
        unsigned long long* __restrict__ fieldGradient,
#endif
#ifdef USE_CUTOFF
        const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize,
        real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ, unsigned int maxTiles, const real4* __restrict__ blockCenter, const unsigned int* __restrict__ interactingAtoms,
#endif
        const float2* __restrict__ dampingAndThole) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];

    // First loop: process tiles that contain exclusions.
    
    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const int2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        AtomData data;
        zeroAtomData(data);
        unsigned int atom1 = x*TILE_SIZE + tgx;
        loadAtomData(data, atom1, posq, inducedDipole, dampingAndThole);
        uint2 covalent = covalentFlags[pos*TILE_SIZE+tgx];
        if (x == y) {
            // This tile is on the diagonal.

            localData[threadIdx.x].pos = data.pos;
            localData[threadIdx.x].inducedDipole = data.inducedDipole;
            localData[threadIdx.x].thole = data.thole;
            localData[threadIdx.x].damp = data.damp;
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                real3 delta = localData[tbx+j].pos-data.pos;
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                int atom2 = y*TILE_SIZE+j;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS){
                    float p = computePScaleFactor(covalent, j);
                    computeOneInteraction(data, localData[tbx+j], delta, p, atom1 == atom2);
                }
            }
        }
        else {
            // This is an off-diagonal tile.

            loadAtomData(localData[threadIdx.x], y*TILE_SIZE+tgx, posq, inducedDipole, dampingAndThole);
            zeroAtomData(localData[threadIdx.x]);
            unsigned int tj = tgx;
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                real3 delta = localData[tbx+tj].pos-data.pos;
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                int atom2 = y*TILE_SIZE+j;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS){
                    float p = computePScaleFactor(covalent, tj);
                    computeOneInteraction(data, localData[tbx+tj], delta, p, false);
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
        }

        // Write results.

        unsigned int offset = x*TILE_SIZE + tgx;
        SAVE_ATOM_DATA(offset, data)
        if (x != y) {
            offset = y*TILE_SIZE + tgx;
            SAVE_ATOM_DATA(offset, localData[threadIdx.x])
        }
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    if (numTiles > maxTiles)
        return; // There wasn't enough memory for the neighbor list.
    int pos = (int) (numTiles > maxTiles ? startTileIndex+warp*(long long)numTileIndices/totalWarps : warp*(long long)numTiles/totalWarps);
    int end = (int) (numTiles > maxTiles ? startTileIndex+(warp+1)*(long long)numTileIndices/totalWarps : (warp+1)*(long long)numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    int pos = (int) (startTileIndex+warp*(long long)numTiles/totalWarps);
    int end = (int) (startTileIndex+(warp+1)*(long long)numTiles/totalWarps);
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;
    
    while (pos < end) {
        bool includeTile = true;

        // Extract the coordinates of this tile.
        
        int x, y;
#ifdef USE_CUTOFF
        x = tiles[pos];
#else
        y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
        x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
            y += (x < y ? -1 : 1);
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        }

        // Skip over tiles that have exclusions, since they were already processed.

        while (skipTiles[tbx+TILE_SIZE-1] < pos) {
            if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                int2 tile = exclusionTiles[skipBase+tgx];
                skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
            }
            else
                skipTiles[threadIdx.x] = end;
            skipBase += TILE_SIZE;            
            currentSkipIndex = tbx;
        }
        while (skipTiles[currentSkipIndex] < pos)
            currentSkipIndex++;
        includeTile = (skipTiles[currentSkipIndex] != pos);
#endif
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.

            AtomData data;
            zeroAtomData(data);
            loadAtomData(data, atom1, posq, inducedDipole, dampingAndThole);
#ifdef USE_CUTOFF
            unsigned int j = interactingAtoms[pos*TILE_SIZE+tgx];
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
            loadAtomData(localData[threadIdx.x], j, posq, inducedDipole, dampingAndThole);
            zeroAtomData(localData[threadIdx.x]);

            // Compute the full set of interactions in this tile.

            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                real3 delta = localData[tbx+tj].pos-data.pos;
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                int atom2 = atomIndices[tbx+tj];
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS)
                    computeOneInteraction(data, localData[tbx+tj], delta, 1, false);
                tj = (tj + 1) & (TILE_SIZE - 1);
            }

            // Write results.

            unsigned int offset = x*TILE_SIZE + tgx;
            SAVE_ATOM_DATA(offset, data)
#ifdef USE_CUTOFF
            offset = atomIndices[threadIdx.x];
#else
            offset = y*TILE_SIZE + tgx;
#endif
            SAVE_ATOM_DATA(offset, localData[threadIdx.x])
        }
        pos++;
    }
}


extern "C" __global__ void recordInducedDipolesForDIIS(const long long* __restrict__ fixedField, const long long* __restrict__ inducedField,
        const real* __restrict__ inducedDipole, const real* __restrict__ labFramePolarizabilities, float* __restrict__ errors,
        real* __restrict__ prevDipoles, real* __restrict__ prevErrors, int iteration, bool recordPrevErrors, real* __restrict__ matrix) {
    extern __shared__ real buffer[];
    const real fieldScale = 1/(real) 0x100000000;
    real sumErrors = 0;
    for (int atom = blockIdx.x*blockDim.x + threadIdx.x; atom < NUM_ATOMS; atom += blockDim.x*gridDim.x) {
        //
        real3 fld = make_real3(fixedField[atom+0*PADDED_NUM_ATOMS] + inducedField[atom+0*PADDED_NUM_ATOMS],
                               fixedField[atom+1*PADDED_NUM_ATOMS] + inducedField[atom+1*PADDED_NUM_ATOMS],
                               fixedField[atom+2*PADDED_NUM_ATOMS] + inducedField[atom+2*PADDED_NUM_ATOMS]);
        int offset = 6*atom;
        for (int component = 0; component < 3; component++) {
            int dipoleIndex = 3*atom+component;
            if (iteration >= MAX_PREV_DIIS_DIPOLES) {
                // We have filled up the buffer for previous dipoles, so shift them all over by one.
                for (int i = 1; i < MAX_PREV_DIIS_DIPOLES; i++) {
                    int index1 = dipoleIndex+(i-1)*NUM_ATOMS*3;
                    int index2 = dipoleIndex+i*NUM_ATOMS*3;
                    prevDipoles[index1] = prevDipoles[index2];
                    if (recordPrevErrors)
                        prevErrors[index1] = prevErrors[index2];
                }
            }
            // Compute the new dipole, and record it along with the error.
            real3 alpha;
            if(component==0)
                alpha = make_real3(labFramePolarizabilities[offset+0], labFramePolarizabilities[offset+1], labFramePolarizabilities[offset+2]);
            else if(component==1)
                alpha = make_real3(labFramePolarizabilities[offset+1], labFramePolarizabilities[offset+3], labFramePolarizabilities[offset+4]);
            else if(component==2)
                alpha = make_real3(labFramePolarizabilities[offset+2], labFramePolarizabilities[offset+4], labFramePolarizabilities[offset+5]);

            real oldDipole = inducedDipole[dipoleIndex];
            real newDipole = fieldScale*dot(alpha, fld);
            int storePrevIndex = dipoleIndex+min(iteration, MAX_PREV_DIIS_DIPOLES-1)*NUM_ATOMS*3;
            prevDipoles[storePrevIndex] = newDipole;
            if (recordPrevErrors)
                prevErrors[storePrevIndex] = newDipole-oldDipole;
            sumErrors += (newDipole-oldDipole)*(newDipole-oldDipole);
        }
    }

    // Sum the errors over threads and store the total for this block.

    buffer[threadIdx.x] = sumErrors;
    __syncthreads();
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        if (threadIdx.x+offset < blockDim.x && (threadIdx.x&(2*offset-1)) == 0) {
            buffer[threadIdx.x] += buffer[threadIdx.x+offset];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        errors[blockIdx.x] = buffer[0];

    if (iteration >= MAX_PREV_DIIS_DIPOLES && recordPrevErrors && blockIdx.x == 0) {
        // Shift over the existing matrix elements.
        for (int i = 0; i < MAX_PREV_DIIS_DIPOLES-1; i++) {
            if (threadIdx.x < MAX_PREV_DIIS_DIPOLES-1)
                matrix[threadIdx.x+i*MAX_PREV_DIIS_DIPOLES] = matrix[(threadIdx.x+1)+(i+1)*MAX_PREV_DIIS_DIPOLES];
            __syncthreads();
        }
    }
}

extern "C" __global__ void computeDIISMatrix(real* __restrict__ prevErrors, int iteration, real* __restrict__ matrix) {
    extern __shared__ real sumBuffer[];
    int j = min(iteration, MAX_PREV_DIIS_DIPOLES-1);
    for (int i = blockIdx.x; i <= j; i += gridDim.x) {
        // All the threads in this thread block work together to compute a single matrix element.

        real sum = 0;
        for (int index = threadIdx.x; index < NUM_ATOMS*3; index += blockDim.x)
            sum += prevErrors[index+i*NUM_ATOMS*3]*prevErrors[index+j*NUM_ATOMS*3];
        sumBuffer[threadIdx.x] = sum;
        __syncthreads();
        for (int offset = 1; offset < blockDim.x; offset *= 2) { 
            if (threadIdx.x+offset < blockDim.x && (threadIdx.x&(2*offset-1)) == 0)
                sumBuffer[threadIdx.x] += sumBuffer[threadIdx.x+offset];
            __syncthreads();
        }
        if (threadIdx.x == 0) {
            matrix[i+MAX_PREV_DIIS_DIPOLES*j] = sumBuffer[0];
            if (i != j)
                matrix[j+MAX_PREV_DIIS_DIPOLES*i] = sumBuffer[0];
        }
    }
}

extern "C" __global__ void solveDIISMatrix(int iteration, const real* __restrict__ matrix, float* __restrict__ coefficients) {
    __shared__ real b[MAX_PREV_DIIS_DIPOLES+1][MAX_PREV_DIIS_DIPOLES+1];
    __shared__ real piv[MAX_PREV_DIIS_DIPOLES+1];
    __shared__ real x[MAX_PREV_DIIS_DIPOLES+1];

    // On the first iteration we don't need to do any calculation.
    
    if (iteration == 0) {
        if (threadIdx.x == 0)
            coefficients[0] = 1;
        return;
    }
    
    // Load the matrix.
    
    int numPrev = min(iteration+1, MAX_PREV_DIIS_DIPOLES);
    int rank = numPrev+1;
    for (int index = threadIdx.x; index < numPrev*numPrev; index += blockDim.x) {
        int i = index/numPrev;
        int j = index-i*numPrev;
        b[i+1][j+1] = matrix[i*MAX_PREV_DIIS_DIPOLES+j];
    }
    for (int i = threadIdx.x; i < rank; i += blockDim.x) {
        b[i][0] = -1;
        piv[i] = i;
    }
    __syncthreads();
    
    // Compute the mean absolute value of the values we just loaded.  We use that for preconditioning it,
    // which is essential for doing the computation in single precision.
    
    if (threadIdx.x == 0) {
        real mean = 0;
        for (int i = 0; i < numPrev; i++)
            for (int j = 0; j < numPrev; j++)
                mean += fabs(b[i+1][j+1]);
        mean /= numPrev*numPrev;
        b[0][0] = 0;
        for (int i = 1; i < rank; i++)
            b[0][i] = -mean;

        // Compute the LU decomposition of the matrix.  This code is adapted from JAMA.
    
        int pivsign = 1;
        for (int j = 0; j < rank; j++) {
            // Apply previous transformations.

            for (int i = 0; i < rank; i++) {
                // Most of the time is spent in the following dot product.

                int kmax = min(i, j);
                real s = 0;
                for (int k = 0; k < kmax; k++)
                    s += b[i][k] * b[k][j];
                b[i][j] -= s;
            }

            // Find pivot and exchange if necessary.

            int p = j;
            for (int i = j+1; i < rank; i++)
                if (abs(b[i][j]) > abs(b[p][j]))
                    p = i;
            if (p != j) {
                int k = 0;
                for (k = 0; k < rank; k++) {
                    real t = b[p][k];
                    b[p][k] = b[j][k];
                    b[j][k] = t;
                }
                k = piv[p];
                piv[p] = piv[j];
                piv[j] = k;
                pivsign = -pivsign;
            }

            // Compute multipliers.

            if ((j < rank) && (b[j][j] != 0))
                for (int i = j+1; i < rank; i++)
                    b[i][j] /= b[j][j];
        }
        for (int i = 0; i < rank; i++)
            if (b[i][i] == 0) {
                // The matrix is singular.
                
                for (int j = 0; j < rank-1; j++)
                    coefficients[j] = 0;
                coefficients[rank-1] = 1;
                return;
            }

        // Solve b*Y = X(piv)
        
        for (int i = 0; i < rank; i++) 
            x[i] = (piv[i] == 0 ? -1 : 0);
        for (int k = 0; k < rank; k++)
            for (int i = k+1; i < rank; i++)
                x[i] -= x[k] * b[i][k];

        // Solve U*X = Y;
        
        for (int k = rank-1; k >= 0; k--) {
            x[k] /= b[k][k];
            for (int i = 0; i < k; i++)
                x[i] -= x[k] * b[i][k];
        }
        
        // Record the coefficients.
        
        real lastCoeff = 1;
        for (int i = 0; i < rank-1; i++) {
            real c = x[i+1]*mean;
            coefficients[i] = c;
            lastCoeff -= c;
        }
        coefficients[rank-1] = lastCoeff;
    }
}

extern "C" __global__ void updateInducedFieldByDIIS(real* __restrict__ inducedDipole, const real* __restrict__ prevDipoles, 
                                                    const float* __restrict__ coefficients, int numPrev) {
    for (int index = blockIdx.x*blockDim.x + threadIdx.x; index < 3*NUM_ATOMS; index += blockDim.x*gridDim.x) {
        real sum = 0;
        for (int i = 0; i < numPrev; i++) {
            sum += coefficients[i]*prevDipoles[i*3*NUM_ATOMS+index];
        }
        inducedDipole[index] = sum;
    }
}

extern "C" __global__ void initExtrapolatedDipoles(real* __restrict__ inducedDipole, real* __restrict__ extrapolatedDipole,
        long long* __restrict__ inducedDipoleFieldGradient) {
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < 3*NUM_ATOMS; index += blockDim.x*gridDim.x) {
        extrapolatedDipole[index] = inducedDipole[index];
    }
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < 6*NUM_ATOMS; index += blockDim.x*gridDim.x) {
        inducedDipoleFieldGradient[index] = 0;
    }
}

extern "C" __global__ void iterateExtrapolatedDipoles(int order, real* __restrict__ inducedDipole, real* __restrict__ extrapolatedDipole,
        long long* __restrict__ inducedDipoleFieldGradient, long long* __restrict__ inducedDipoleField,
        real* __restrict__ extrapolatedDipoleField, real* __restrict__ extrapolatedDipoleFieldGradient, const real* __restrict__ labFramePolarizabilities) {
    const real fieldScale = 1/(real) 0x100000000;
    for (int atom = blockIdx.x*blockDim.x + threadIdx.x; atom < NUM_ATOMS; atom += blockDim.x*gridDim.x) {
        int offset = 6*atom;
        real3 fld = make_real3(inducedDipoleField[atom+0*PADDED_NUM_ATOMS],
                               inducedDipoleField[atom+1*PADDED_NUM_ATOMS],
                               inducedDipoleField[atom+2*PADDED_NUM_ATOMS]);
        for (int component = 0; component < 3; component++) {
            // Compute the new dipole, and record it along with the error.
            real3 alpha;
            if(component==0)
                alpha = make_real3(labFramePolarizabilities[offset+0], labFramePolarizabilities[offset+1], labFramePolarizabilities[offset+2]);
            else if(component==1)
                alpha = make_real3(labFramePolarizabilities[offset+1], labFramePolarizabilities[offset+3], labFramePolarizabilities[offset+4]);
            else if(component==2)
                alpha = make_real3(labFramePolarizabilities[offset+2], labFramePolarizabilities[offset+4], labFramePolarizabilities[offset+5]);
            real value = dot(alpha,fld)*fieldScale;
            inducedDipole[3*atom+component] = value;
            extrapolatedDipole[order*3*NUM_ATOMS+3*atom+component] = value;
        }

    }
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < 3*NUM_ATOMS; index += blockDim.x*gridDim.x) {
        int index2 = (order-1)*3*NUM_ATOMS+index;
        int atom = index/3;
        int component = index%3;
        extrapolatedDipoleField[index2] = fieldScale*inducedDipoleField[atom+component*PADDED_NUM_ATOMS];
    }
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < 6*NUM_ATOMS; index += blockDim.x*gridDim.x) {
        int index2 = (order-1)*6*NUM_ATOMS+index;
        extrapolatedDipoleFieldGradient[index2] = fieldScale*inducedDipoleFieldGradient[index];
    }
}

extern "C" __global__ void computeExtrapolatedDipoles(real* __restrict__ inducedDipole, real* __restrict__ extrapolatedDipole) {
    real coeff[] = {EXTRAPOLATION_COEFFICIENTS_SUM};
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < 3*NUM_ATOMS; index += blockDim.x*gridDim.x) {
        real sum = 0;
        for (int order = 0; order < MAX_EXTRAPOLATION_ORDER; order++) {
            sum += extrapolatedDipole[order*3*NUM_ATOMS+index]*coeff[order];
        }
        inducedDipole[index] = sum;
    }
}

extern "C" __global__ void addExtrapolatedFieldGradientToForce(long long* __restrict__ forceBuffers,
        long long* __restrict__ torqueBuffers,
        const float2* __restrict__ dampingAndThole,
        real* __restrict__ extrapolatedDipole,
        real* __restrict__ extrapolatedDipoleField, real* __restrict__ extrapolatedDipoleFieldGradient ) {
    real coeff[] = {EXTRAPOLATION_COEFFICIENTS_SUM};
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < NUM_ATOMS; atom += blockDim.x*gridDim.x) {
        real fx = 0, fy = 0, fz = 0;
        real tx = 0, ty = 0, tz = 0;
        bool isAnisotropic = dampingAndThole[atom].x < 0 ? false : true;
        for (int l = 0; l < MAX_EXTRAPOLATION_ORDER-1; l++) {
            int index1 = 3*(l*NUM_ATOMS+atom);
            real dipole[] = {extrapolatedDipole[index1], extrapolatedDipole[index1+1], extrapolatedDipole[index1+2]};
            for (int m = 0; m < MAX_EXTRAPOLATION_ORDER-1-l; m++) {
                int index2 = 6*(m*NUM_ATOMS+atom);
                int index3 = 3*(m*NUM_ATOMS+atom);
                real scale = coeff[l+m+1]*ENERGY_SCALE_FACTOR;
                real fieldGradient[] = {extrapolatedDipoleFieldGradient[index2], extrapolatedDipoleFieldGradient[index2+1], extrapolatedDipoleFieldGradient[index2+2],
                                   extrapolatedDipoleFieldGradient[index2+3], extrapolatedDipoleFieldGradient[index2+4], extrapolatedDipoleFieldGradient[index2+5]};
                real dipoleField[] = {extrapolatedDipoleField[index3], extrapolatedDipoleField[index3+1], extrapolatedDipoleField[index3+2]};
                fx += scale*(dipole[0]*fieldGradient[0] + dipole[1]*fieldGradient[3] + dipole[2]*fieldGradient[4]);
                fy += scale*(dipole[0]*fieldGradient[3] + dipole[1]*fieldGradient[1] + dipole[2]*fieldGradient[5]);
                fz += scale*(dipole[0]*fieldGradient[4] + dipole[1]*fieldGradient[5] + dipole[2]*fieldGradient[2]);
                tx += scale*(dipole[1]*dipoleField[2] - dipole[2]*dipoleField[1]);
                ty += scale*(dipole[2]*dipoleField[0] - dipole[0]*dipoleField[2]);
                tz += scale*(dipole[0]*dipoleField[1] - dipole[1]*dipoleField[0]);
            }
        }
        forceBuffers[atom] += (long long) (fx*0x100000000);
        forceBuffers[atom+PADDED_NUM_ATOMS] += (long long) (fy*0x100000000);
        forceBuffers[atom+PADDED_NUM_ATOMS*2] += (long long) (fz*0x100000000);
        torqueBuffers[atom] += (long long) (isAnisotropic?tx*0x100000000:0);
        torqueBuffers[atom+PADDED_NUM_ATOMS] += (long long) (isAnisotropic?ty*0x100000000:0);
        torqueBuffers[atom+PADDED_NUM_ATOMS*2] += (long long) (isAnisotropic?tz*0x100000000:0);
    }
}
